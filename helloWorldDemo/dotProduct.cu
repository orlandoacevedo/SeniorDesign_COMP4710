#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void arrayMult(float *a, float *b, float *result, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        result[idx] = a[idx] * b[idx];
    }
}



int main(){
    float *vector_a, *vector_b;
    float *dev_a, *dev_b, *dev_result;
    float *result;

    int N = 16;
    size_t size = N * sizeof(*vector_a);
    vector_a = (float *) malloc(size);
    vector_b = (float *) malloc(size);
    result = (float *) malloc(size);

    hipMalloc( (void **) &dev_a, size);
    hipMalloc( (void **) &dev_b, size);
    hipMalloc( (void **) &dev_result, size);
    int i;

    for (i = 0; i < N; i++){
        vector_a[i] = 2.f;
        vector_b[i] = 4.f;
    }

    hipMemcpy(dev_a, vector_a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vector_b, sizeof(float) * N, hipMemcpyHostToDevice);

    int blockSize = 4;
    int nBlocks = 4;

    arrayMult <<<nBlocks, blockSize>>> (dev_a, dev_b, dev_result, N);

    hipMemcpy(result, dev_result, sizeof(float) * N, hipMemcpyDeviceToHost);

    float dotProduct = 0;
    for (i = 0; i < N; i++){
        dotProduct = dotProduct + result[i];
    }
    printf("Vector_A: ");
    for (i = 0; i < N; i++){
        printf("%f,", vector_a[i]);
    }
    printf("\nVector_B: ");
    for (i = 0; i < N; i++){
        printf("%f,", vector_b[i]);
        
    }
    
    printf("\nResult = %f\n", dotProduct);
    
}
