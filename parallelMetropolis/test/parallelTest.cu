#include "hip/hip_runtime.h"
#include "parallelTest.cuh"

/**
    Wrapper function that will call the global function used to 
    test the functions that calculate indexes in the half array
    used to hold the energies.
*/
void setupGetIndexTest(){
    int numberOfBlocks = 3;
    int threadsPerBlock = 2;
    int totalTests = numberOfBlocks * threadsPerBlock;

    int *xValues;
    int *yValues;
    int *yValues_device;
    int *xValues_device;
    
    size_t xSize = totalTests * sizeof(int);
    
    yValues = (int *) malloc(xSize);
    xValues = (int *)malloc(xSize);
    hipMalloc((void **) &yValues_device, xSize);
    hipMalloc((void **) &xValues_device, xSize);
    
    testGetXKernel <<<numberOfBlocks, threadsPerBlock>>>(xValues_device, totalTests);

    hipMemcpy(xValues, xValues_device, xSize, hipMemcpyDeviceToHost);

    assert(xValues[0] == 1);
    assert(xValues[1] == 2);
    assert(xValues[2] == 2);
    assert(xValues[3] == 3);
    assert(xValues[4] == 3);
    assert(xValues[5] == 3);

    printf("getXFromIndex Correct\n");

    //test getYFromIndex)
    testGetYKernel <<<numberOfBlocks, threadsPerBlock>>> (xValues_device,
            yValues_device, totalTests);

    hipMemcpy(yValues, yValues_device, xSize, hipMemcpyDeviceToHost);

    assert(yValues[0] == 0);
    assert(yValues[1] == 0);
    assert(yValues[2] == 1);
    assert(yValues[3] == 0);
    assert(yValues[4] == 1);
    assert(yValues[5] == 2);

    printf("getYFromIndex Correct.\n");

    hipFree(xValues_device);
    hipFree(yValues_device);
    free(yValues);
    free(xValues);
}

bool compareDouble(double a, double b, double limit){
    if((a - b) / b < limit)
        return true;
    else
        return false;
}


/**
  wrapper function for the __device__ makePeriodic function
*/
void setupMakePeriodic(){
    srand(time(NULL));
    int numberOfTests = 128;
    double *box;;
    
    double *inputs_host;
    double *inputs_device;
    double *outputs_host;
    double *dev_box;
    size_t inputSize = sizeof(double) * numberOfTests;

    box = (double *) malloc(sizeof(double));
    *box = 10.0;
    inputs_host = (double *) malloc(inputSize);
    outputs_host = (double *) malloc(inputSize);
    hipMalloc((void **) &inputs_device, inputSize);
    hipMalloc((void **) &dev_box, sizeof(double));
    
    //generate random numbers
    for(int i = 0; i < numberOfTests; i++){
        inputs_host[i] = ((double) (rand() % 100));
    }

    //copy data to device
    hipMemcpy(inputs_device, inputs_host, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_box, box, sizeof(double), hipMemcpyHostToDevice);
    
    int threadsPerBlock = numberOfTests / 2;
    int blocks = numberOfTests / threadsPerBlock +
        (numberOfTests % threadsPerBlock == 0 ? 0 : 1);

    testMakePeriodicKernel <<< blocks, threadsPerBlock >>> (inputs_device,
            dev_box, numberOfTests);

    hipMemcpy(outputs_host, inputs_device, inputSize, hipMemcpyDeviceToHost);

    //check that values are the same as known correct function
    for(int i = 0; i < numberOfTests; i++){
        double test_output = make_periodic(inputs_host[i], *box);
        assert(outputs_host[i] == test_output);
    }

    printf("makePeriodic passed Tests\n");

    free(inputs_host);
    free(outputs_host);
    hipFree(inputs_device);


}


void testWrapBox(){
 srand(time(NULL));
    int numberOfTests = 128;
    double box;
    
    double *testDoubles;
    size_t inputSize = sizeof(double) * numberOfTests;

    box = 10.f;
    testDoubles = (double *) malloc(inputSize);
    
    //generate random numbers
    for(int i = 0; i < numberOfTests; i++){
        testDoubles[i] = ((double) (rand() % 100));
    }

     //check that values are the same as known correct function
    for(int i = 0; i < numberOfTests; i++){
        double test_output = wrap_into_box(testDoubles[i], box);
        assert(wrapBox(testDoubles[i], box) == test_output);
    }

    printf("wrapBox passed Tests\n");

    free(testDoubles);


}

void setupCalc_lj(){
    double kryptonSigma = 3.624;
    double kryptonEpsilon = 0.317;
    int numberOfAtoms = 2;

    Atom *atoms = new Atom[numberOfAtoms];
    double *energy = (double *) malloc(sizeof(double));
    *energy = 1000.f;
    Atom *atoms_device;
    Environment *enviro_device;
    double *energy_device;

    hipMalloc((void **) &atoms_device, sizeof(Atom) * numberOfAtoms);
    hipMalloc((void **) &enviro_device, sizeof(Environment));
    hipMalloc((void **) &energy_device, sizeof(double));

    Environment stableEnviro = createEnvironment(10, 10, 10, .5,
            298.15, numberOfAtoms);

    Environment *enviro = &stableEnviro;
    generatePoints(atoms, enviro);
    atoms[0].sigma = kryptonSigma;
    atoms[0].epsilon = kryptonEpsilon; 
    atoms[1].sigma = kryptonSigma;
    atoms[1].epsilon = kryptonEpsilon;

    hipMemcpy(atoms_device, atoms, sizeof(Atom) * numberOfAtoms, hipMemcpyHostToDevice);
    hipMemcpy(enviro_device, enviro, sizeof(Environment), hipMemcpyHostToDevice);

    testCalcLJ<<<1,1>>>(atoms_device, enviro_device, energy_device);

    hipMemcpy(energy, energy_device, sizeof(double), hipMemcpyDeviceToHost);

    double baseEnergy = calculate_energy(atoms, enviro);
    assert((int)(*energy * pow(10.f, 6.f)) == (int)( baseEnergy * pow(10.f,6.f))); 
    printf("\nparallelEnergy = %2.10f\nlinearEnergy = %2.10f\n", *energy, baseEnergy); 
    printf("Calc_lj is correct\n");
    free(atoms);
    free(energy);
    hipFree(atoms_device);
    hipFree(enviro_device);
    hipFree(energy_device);
}


void testGeneratePoints(){
    //init atoms, environment
    int numberOfAtoms = 1000;
    Atom *atoms = (Atom *) malloc(numberOfAtoms * sizeof(Atom));

    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, i, 1.1*i, 1.2*i);
    }
    Environment enviro = createEnvironment(10.0, 20.0, 35.0, 1.0, 298.15, numberOfAtoms);

    generatePoints(atoms, &enviro);

    //assert that all atoms positions are in range of the box
    for (int i = 0; i < numberOfAtoms; i++){
        double dim_x = atoms[i].x;
        double dim_y = atoms[i].y;
        double dim_z = atoms[i].z;

        assert(dim_x >= i && dim_x <= (enviro.x + i) &&
               dim_y >= (1.1 * i) && dim_y <= (enviro.y + 1.1 * i) &&
               dim_z >= (1.2 * i) && dim_z <= (enviro.z + 1.2 * i));
    }
    printf("testGeneratePoints successful.\n");

    free(atoms);
}

void testCalcEnergy(){
    // the sigma value of krypton used in the LJ simulation
    double kryptonSigma = 3.624;
    // the epsilon value of krypton used in the LJ simulation
    double kryptonEpsilon = 0.317;

    struct timeval le_tvBegin, le_tvEnd, pl_tvBegin, pl_tvEnd;

    //Generate enviorment and atoms
    int numberOfAtoms = 1000;
    Environment stableEnviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 298.15, numberOfAtoms);

    Environment *enviro = &stableEnviro;

    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, 0.0, 0.0, 0.0, kryptonSigma, kryptonEpsilon);
    }

    generatePoints(atoms, enviro);
    
    //calculate energy linearly
    gettimeofday(&le_tvBegin,NULL); //start clock for execution time

    double te_linear = calculate_energy(atoms, enviro);

    gettimeofday(&le_tvEnd,NULL); //stop clock for execution time
    long le_runTime = timevaldiff(&le_tvBegin,&le_tvEnd); //get difference in time in milli seconds

    //calculate energy in parallel
    gettimeofday(&pl_tvBegin,NULL); //start clock for execution time

    double te_parallel =  calcEnergyWrapper(atoms, enviro);	 

    gettimeofday(&pl_tvEnd,NULL); //start clock for execution time
    long pl_runTime = timevaldiff(&pl_tvBegin,&pl_tvEnd); //get difference in time in milli seconds


    //Print out Results
    printf("Number of elements: %d\n", numberOfAtoms);
    printf("Linear Total Energy:   %f \n", te_linear);
    printf("In %d ms\n", le_runTime);
    printf("Parallel Total Energy: %f \n", te_parallel);
    printf("In %d ms\n", pl_runTime);
    assert(compareDouble(te_linear, te_parallel, .05));
    printf("testCalcEnergy successful.\n");

    
}

void testCalcEnergyWithMolecules(){
    // the sigma value of krypton used in the LJ simulation
    double kryptonSigma = 3.624;
    // the epsilon value of krypton used in the LJ simulation
    double kryptonEpsilon = 0.317;

    struct timeval le_tvBegin, le_tvEnd, pl_tvBegin, pl_tvEnd;

    //Generate enviorment and atoms
    int numberOfAtoms = 100;
    Environment stableEnviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 298.15, numberOfAtoms);

    Environment *enviro = &stableEnviro;

    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, 0.0, 0.0, 0.0, kryptonSigma, kryptonEpsilon);
    }
    enviro->numOfMolecules = numberOfAtoms;
    generatePoints(atoms, enviro);
    Molecule *molecules;
    molecules = (Molecule *)malloc(sizeof(Molecule) * numberOfAtoms);
    for(int i = 0; i < numberOfAtoms; i++){
        molecules[i].numOfAtoms = 1;
        molecules[i].atoms = (Atom *)malloc(sizeof(Atom));
        molecules[i].atoms[0] = atoms[i];
    }

    //calculate energy linearly
    gettimeofday(&le_tvBegin,NULL); //start clock for execution time

    double te_linear = calculate_energy(atoms, enviro);

    gettimeofday(&le_tvEnd,NULL); //stop clock for execution time
    long le_runTime = timevaldiff(&le_tvBegin,&le_tvEnd); //get difference in time in milli seconds

    //calculate energy in parallel
    gettimeofday(&pl_tvBegin,NULL); //start clock for execution time

    double te_parallel =  calcEnergyWrapper(molecules, enviro);	 

    gettimeofday(&pl_tvEnd,NULL); //start clock for execution time
    long pl_runTime = timevaldiff(&pl_tvBegin,&pl_tvEnd); //get difference in time in milli seconds


    //Print out Results
    printf("Number of elements: %d\n", numberOfAtoms);
    printf("Linear Total Energy:   %f \n", te_linear);
    printf("In %d ms\n", le_runTime);
    printf("Parallel Total Energy: %f \n", te_parallel);
    printf("In %d ms\n", pl_runTime);
    assert(compareDouble(te_linear, te_parallel, .05));
    printf("testCalcEnergyWithMolecules successful.\n");

    
}

void testGetMoleculeFromIDWrapper(){
    int numberOfAtoms = 11;
    int numberOfMolecules = 3;
    
    Atom *atoms;
    Molecule *molecules;
    Environment enviro;
    int *answers;

    Atom *atoms_device;
    Molecule *molecules_device;
    int *answers_device;

    enviro.numOfAtoms = numberOfAtoms;
    enviro.numOfMolecules = numberOfMolecules;

    atoms = (Atom *)malloc(sizeof(Atom) * numberOfAtoms);
    molecules = (Molecule *)malloc(sizeof(Molecule) *numberOfMolecules);
    answers = (int *)malloc(sizeof(int) * numberOfAtoms);

    hipMalloc((void **) &atoms_device, sizeof(Atom) * numberOfAtoms);
    hipMalloc((void **) &molecules_device, sizeof(Molecule) * numberOfMolecules);
    hipMalloc((void **) &answers_device, sizeof(int) * numberOfAtoms);

    enviro.numOfAtoms = numberOfAtoms;
    enviro.numOfMolecules = numberOfMolecules;
    
    for(int i = 0; i < numberOfAtoms; i++){
        atoms[i].id = i;
    }
    molecules[0].id = 0;
    molecules[1].id = 2;
    molecules[2].id = 6;


    hipMemcpy(atoms_device, atoms, sizeof(Atom) * numberOfAtoms, hipMemcpyHostToDevice);
    hipMemcpy(molecules_device, molecules, sizeof(Molecule) * numberOfMolecules, hipMemcpyHostToDevice);

    int numberOfBlocks = 1;
    int threadsPerBlock = 128;
    testGetMoleculeFromID<<<numberOfBlocks,threadsPerBlock>>>(atoms_device,
            molecules_device, enviro, numberOfAtoms, answers_device);
   
    hipMemcpy(answers, answers_device, sizeof(int) * numberOfAtoms, hipMemcpyDeviceToHost);

    assert(answers[0] == 0);
    assert(answers[1] == 0);
    assert(answers[2] == 2);
    assert(answers[3] == 2);
    assert(answers[4] == 2);
    assert(answers[5] == 2);
    assert(answers[6] == 6);
    assert(answers[7] == 6);
    assert(answers[8] == 6);
    assert(answers[9] == 6);
    assert(answers[10] == 6);
   
    printf("getMoleculeFromID passed tests\n");

    free(atoms);
    free(molecules);
    free(answers);

    hipFree(atoms_device);
    hipFree(molecules_device);
    hipFree(answers_device);


}


void testCalcBlendingWrapper(){
    double *d1, *d2, *d1_device, *d2_device, *answers, *answers_device;
    int numberOfTests = 5;
    size_t doubleSize = sizeof(double) * numberOfTests;

    d1 = (double *)malloc(doubleSize);
    d2 = (double *)malloc(doubleSize);
    answers = (double *)malloc(doubleSize);
    
    hipMalloc((void **) &d1_device, doubleSize);
    hipMalloc((void **) &d2_device, doubleSize);
    hipMalloc((void **) &answers_device, doubleSize);

    d1[0] = 0.f;
    d2[0] = 0.f;

    d1[1] = 4.5;
    d2[1] = 2.32;
    
    d1[2] = 52.34;
    d2[2] = 5.f;


    d1[3] = 1.f;
    d2[3] = 7.f;

    d1[4] = 34.56;
    d2[4] = 12.7;
    
    hipMemcpy(d1_device, d1, doubleSize, hipMemcpyHostToDevice);
    hipMemcpy(d2_device, d2, doubleSize, hipMemcpyHostToDevice);

    int blocks = 1;
    int threadsPerBlock = 64;

    testCalcBlending <<<blocks, threadsPerBlock>>>(d1_device, d2_device, answers_device, numberOfTests);

    hipMemcpy(answers, answers_device, doubleSize, hipMemcpyDeviceToHost);

    for(int i = 0 ; i < numberOfTests; i++){
        double expected = sqrt(d1[i] * d2[i]);
        assert(answers[i] / sqrt(d1[i] * d2[i]) < 0.01 || answers[i] == expected);
    }

    printf("calcBlending passed tests.\n");
    
    free(d1);
    free(d2);
    free(answers);
    hipFree(d1_device);
    hipFree(d2_device);
    hipFree(answers_device);
}

void testGetFValueWrapper(){
    Environment *enviro, *dev_enviro;
    Molecule *molecules, *dev_molecules;
    Atom *mol1_atoms, *mol2_atoms, *atom1List, *atom2List, *dev_atom1List, *dev_atom2List;
    double *fvalues, *dev_fvalues;
    Bond *mol1_bonds, *blankBonds;
    Angle *blankAngles;
    Dihedral *blankDihedrals;

    int numberOfTests = 4;

    Environment stable_enviro = createEnvironment(5.0,5.0,5.0,1.0,270.0,5);
    enviro = &stable_enviro;
    mol1_atoms = (Atom *)malloc(sizeof(Atom)*4);
    mol2_atoms = (Atom *)malloc(sizeof(Atom));
    atom1List = (Atom *)malloc(sizeof(Atom)*4);
    atom2List = (Atom *)malloc(sizeof(Atom)*4);

    fvalues = (double *)malloc(sizeof(double)*4);
    
    for (int i = 0; i < 4; i++){
        mol1_atoms[i] = createAtom(i+1,1.0,1.0,1.0);
    }
    for (int i = 0; i < 4; i++){
        atom1List[i] = mol1_atoms[0];
        if (i < 3)
            atom2List[i] = mol1_atoms[i+1];
    }

    mol2_atoms[0] = createAtom(5,1.0,1.0,1.0);
    atom2List[4] = mol2_atoms[0];

    mol1_bonds = (Bond *)malloc(sizeof(Bond)*3);
    for (int i = 0; i < 3; i++){
        mol1_bonds[i] = createBond(i+1,i+2, 0.5, false);
    }

    molecules = (Molecule *)malloc(sizeof(Molecule)*2);
    molecules[0] = createMolecule(1, mol1_atoms, blankAngles, mol1_bonds, blankDihedrals, 4, 0, 3, 0);
    molecules[1] = createMolecule(5, mol2_atoms, blankAngles, blankBonds, blankDihedrals, 1, 0, 0, 0);

    hipMalloc((void **) &dev_enviro, sizeof(Environment));
    hipMalloc((void **) &dev_molecules, sizeof(Molecule)*2);
    hipMalloc((void **) &dev_atom1List, sizeof(Atom)*4);
    hipMalloc((void **) &dev_atom2List, sizeof(Atom)*4);
    hipMalloc((void **) &dev_fvalues, sizeof(double)*4);

    hipMemcpy(dev_enviro, enviro, sizeof(Environment), hipMemcpyHostToDevice);
    hipMemcpy(dev_molecules, molecules, sizeof(Molecule)*2, hipMemcpyHostToDevice);
    hipMemcpy(dev_atom1List, atom1List, sizeof(Atom)*4, hipMemcpyHostToDevice);
    hipMemcpy(dev_atom2List, atom2List, sizeof(Atom)*4, hipMemcpyHostToDevice);

    int blocks = 1;
    int threadsPerBlock = 64;

    testGetFValue <<<blocks, threadsPerBlock>>>(dev_atom1List, dev_atom2List, dev_molecules, dev_enviro, dev_fvalues, numberOfTests);

    hipMemcpy(fvalues, dev_fvalues, sizeof(double)*4, hipMemcpyDeviceToHost);

    double *expected = (double *)malloc(sizeof(double)*4);
    expected[0] = 0.0;
    expected[1] = 0.0;
    expected[2] = 0.5;
    expected[3] = 1.0;
    for(int i = 0 ; i < numberOfTests; i++){
        assert(expected[i] == fvalues[i]);
    }

    printf("testGetFValue passed tests.\n");
   
    free(mol1_atoms);
    free(mol2_atoms);
    free(atom1List);
    free(atom2List);
    free(fvalues);
    free(molecules);
    hipFree(dev_enviro);
    hipFree(dev_molecules);
    hipFree(dev_atom1List);
    hipFree(dev_atom2List);
    hipFree(dev_fvalues);
}

Atom findMaxRotation(Atom pivot, Atom toRotate, double rotation){
    toRotate.x -= pivot.x;
    toRotate.y -= pivot.y;
    toRotate.z -= pivot.z;

    rotateAboutX(toRotate, rotation);
    rotateAboutY(toRotate, rotation);
    rotateAboutZ(toRotate, rotation);

    toRotate.x += pivot.x;
    toRotate.y += pivot.y;
    toRotate.z += pivot.z;

    return toRotate;
}

void testRotateMolecule(){
    srand(time(NULL));
    
    //Testing on a molecule that is not totaly unlike water
    double bondDistance = 0.9584; // angstroms
    double maxRotation = 10.0; // degrees
    int numOfAtoms = 3;
    int numOfAngles = 1;
    int numOfBonds = 2;
    int numOfDihedrals = 0;

    Atom oxygen = createAtom(1, 0, 0, 0);
    Atom hydrogen1 = createAtom(2, 0, bondDistance, 0);
    Atom hydrogen2 = createAtom(3, bondDistance, 0, 0);

    Atom *atoms = (Atom *)malloc(sizeof(Atom) * 3);
    atoms[0] = oxygen;
    atoms[1] = hydrogen1;
    atoms[2] = hydrogen2;
    
    vector<Atom> atomVector;
    atomVector.push_back(oxygen);
    atomVector.push_back(hydrogen1);
    atomVector.push_back(hydrogen2);
    Bond b1 = createBond(1,2, bondDistance, false);
    Bond b2 = createBond(1,3, bondDistance, false);
    
    Bond *bonds = (Bond *)malloc(sizeof(Bond) * 2);
    bonds[0] = b1;
    bonds[1] = b2;

    Angle a1 = createAngle(2,3,90,false);
    Angle *angles = (Angle *)malloc(sizeof(Angle));
    angles[0] = a1;

    Dihedral *dihedrals = (Dihedral *)malloc(sizeof(Dihedral) * 0);

    Molecule molec;
    molec = createMolecule(1, atoms, angles, bonds, dihedrals,
            numOfAtoms, numOfAngles, numOfBonds, numOfDihedrals);
    
    int testNumber = 10;
    
    printf("Testing rotateMolecule\n");

    for(int i = 0 ; i < testNumber; i++){
        int roAtom = 1;
        //pick atom to rotate about.  Cycle through all of them
        Atom toRotate = atoms[1];
        
        
        rotateMolecule(molec, toRotate, maxRotation);
        
        //test that rotation is within limit
        Atom newAtom1 = atoms[2];
        Atom origAtom1 = getAtom(atomVector, newAtom1.id);
        
        double angleChange1 = getAngle(newAtom1, toRotate, origAtom1);
        printf("Atom1 angle change = %f\n", angleChange1);

        Atom newAtom2 = atoms[0];
        Atom origAtom2 = getAtom(atomVector, newAtom2.id);
        double angleChange2 = getAngle(newAtom2, toRotate, origAtom2);
        
        printf("Atom2 angle change = %f\n", angleChange2);
        
        Atom maxAtom1 = findMaxRotation(toRotate, newAtom1, maxRotation);
        Atom maxAtom2 = findMaxRotation(toRotate, newAtom2, maxRotation);
        double maxAngle1 = getAngle(maxAtom1, toRotate, origAtom1);
        double maxAngle2 = getAngle(maxAtom2, toRotate, origAtom2);
       
       /** 
        printf("maxRotation = %f", getAngle(maxAtom1, toRotate, origAtom1));
        printf("atom1 = %f, %f, %f\n", origAtom1.x, origAtom1.y, origAtom1.z);
        printf("atom1 = %f, %f, %f\n", newAtom1.x, newAtom1.y, newAtom1.z);
        printf("atom2 = %f, %f, %f\n", origAtom2.x, origAtom2.y, origAtom2.z);
        printf("atom2 = %f, %f, %f\n", newAtom2.x, newAtom2.y, newAtom2.z);
        printf("rotate = %f %f %f\n", toRotate.x, toRotate.y, toRotate.z);
        printf("rotate = %f %f %f\n", atoms[1].x, atoms[1].y, atoms[1].z);
        */
        assert(angleChange1 <= maxAngle1);
        assert(angleChange2 <= maxAngle2);


        //reset atoms
        molec.atoms[0] = oxygen; 
        molec.atoms[1] = hydrogen1;
        molec.atoms[2] = hydrogen2;
    }
    /*
   */ 
    printf("rotateMolecule passed tests.\n");
}

void testCalcChargeWrapper(){
    
    printf("Testing calcCharge()\n");
    
    int numberOfTests = 10;
    
    // data on the host
    Atom *atoms1_h;
    Atom *atoms2_h;
    Environment *enviro_h;
    double *answers_h;

    // data on the device
    Atom *atoms1_d;
    Atom *atoms2_d;
    Environment *enviro_d;
    double *answers_d;

    // get sizes of data
    size_t atomSize = sizeof(Atom) * numberOfTests;
    size_t enviroSize = sizeof(Environment);
    size_t answerSize = sizeof(double) * numberOfTests;

    // mallocate on host
    atoms1_h = (Atom *)malloc(atomSize);
    atoms2_h = (Atom *)malloc(atomSize);
    enviro_h = (Environment *)malloc(enviroSize);
    answers_h = (double *) malloc(answerSize);

    // mallocate on device
    hipMalloc((void **) &atoms1_d, atomSize);
    hipMalloc((void **) &atoms2_d, atomSize);
    hipMalloc((void **) &enviro_d, enviroSize);
    hipMalloc((void **) &answers_d, answerSize);

    double xSize = 10;
    double ySize = xSize;
    double zSize = ySize;

    //generate atoms for test
    srand(time(NULL));
    for(int i = 0; i < numberOfTests; i++){
        atoms1_h[i].x = (double) rand() / (double) RAND_MAX * xSize;
        atoms2_h[i].x = (double) rand() / (double) RAND_MAX * xSize;
        
        atoms1_h[i].y = (double) rand() / (double) RAND_MAX * ySize;
        atoms2_h[i].y = (double) rand() / (double) RAND_MAX * ySize;
        
        atoms1_h[i].z = (double) rand() / (double) RAND_MAX * zSize;
        atoms2_h[i].z = (double) rand() / (double) RAND_MAX * zSize;
   
        atoms1_h[i].charge = (double) rand() / (double) RAND_MAX * 2 - 1;
        atoms2_h[i].charge = (double) rand() / (double) RAND_MAX * 2 - 1; 
    }

    enviro_h->x = xSize;
    enviro_h->y = ySize;
    enviro_h->z = zSize;
    enviro_h->numOfAtoms = numberOfTests;

    //transfer data to the device
    hipMemcpy(atoms1_d, atoms1_h, atomSize, hipMemcpyHostToDevice);
    hipMemcpy(atoms2_d, atoms2_h, atomSize, hipMemcpyHostToDevice);
    hipMemcpy(enviro_d, enviro_h, enviroSize, hipMemcpyHostToDevice);

    //call test function
    int numOfBlocks = 1;
    int threadsPerBlock = 64;
    
    testCalcCharge<<<numOfBlocks, threadsPerBlock>>>(atoms1_d, atoms2_d, answers_d, enviro_d);

    //transfer answers from device to host
    hipMemcpy(answers_h, answers_d, answerSize, hipMemcpyDeviceToHost);

    //TEST ANSWERS
    for(int i = 0; i < numberOfTests; i++){
        double expected = calc_charge(atoms1_h[i], atoms2_h[i], *enviro_h);
        assert((expected - answers_h[i]) / expected < .01);
    }

    printf("calcCharge passed tests.\n");

    free(atoms1_h);
    free(atoms2_h);
    free(enviro_h);
    free(answers_h);

    hipFree(atoms1_d);
    hipFree(atoms2_d);
    hipFree(enviro_d);
    hipFree(answers_d);
}

int main(){
    testRotateMolecule();
    testCalcChargeWrapper();
    testCalcBlendingWrapper();
    testGetMoleculeFromIDWrapper();
    testWrapBox();
    setupCalc_lj();
    setupGetIndexTest();
    setupMakePeriodic();
    testGeneratePoints();
    testCalcEnergy();
    testCalcEnergyWithMolecules();
    return 0;
}

