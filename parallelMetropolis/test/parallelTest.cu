#include "parallelTest.cuh"

void setupGetXFromIndex(){
    int numberOfBlocks = 3;
    int threadsPerBlock = 2;

    int *xValues;
    int *xValues_device;
    
    size_t xSize = numberOfBlocks * threadsPerBlock * sizeof(int);
    xValues = (int *)malloc(xSize);
    hipMalloc((void **) &xValues_device, xSize);
    
    testGetXKernel <<<numberOfBlocks, threadsPerBlock>>>(xValues_device);

    hipMemcpy(xValues, xValues_device, xSize, hipMemcpyDeviceToHost);

    assert(xValues[0] == 1);
    assert(xValues[1] == 2);
    assert(xValues[2] == 2);
    assert(xValues[3] == 3);
    assert(xValues[4] == 3);
    assert(xValues[5] == 3);

    printf("getXFromIndex Correct\n");

    hipFree(xValues_device);
    free(xValues);
}


void setupGetYFromIndex(){
    //TODO
}


void setupMakePeriodic(){
    //TODO
}



void setupWrapBox(){
    //TODO
}


void setupCalc_lj(){
    //TODO
}


void testGeneratePoints(){
    //init atoms, environment
    int numberOfAtoms = 10;
    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, -1.0, -1.0, -1.0);
    }
    Environment stableEnviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 122.0, numberOfAtoms);

    Environment *enviro = &stableEnviro;

    generatePoints(atoms, enviro);

    //assert that all atoms positions are in range of the box
    for (int i = 0; i < numberOfAtoms; i++){
        double dim_x = atoms[i].x;
        double dim_y = atoms[i].y;
        double dim_z = atoms[i].z;
        
        printf("%f, %f, %f\n", dim_x, dim_y, dim_z);

        assert(dim_x >= 0.0 && dim_x <= enviro->x &&
               dim_y >= 0.0 && dim_y <= enviro->y &&
               dim_z >= 0.0 && dim_z <= enviro->z);
    }
    printf("testGeneratePoints successful.");
}

void testCalcEnergy(){
    //TODO    
}

int main(){
    testGeneratePoints();    
    return 0;
}
