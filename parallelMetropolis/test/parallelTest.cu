#include "hip/hip_runtime.h"
#include "parallelTest.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

void setupGetXFromIndex(){
    //TODO
}
__global__ void testGetXKernel(){
    //TODO
}

void setupGetYFromIndex(){
    //TODO
}
__global__ void testGetYKernel(){
    //TODO
}

void setupMakePeriodic(){
    //TODO
}

__global__ void testMakePeriodicKernel(){
    //TODO
}

void setupWrapBox(){
    //TODO
}
__global__ void testWrapBoxKernel(){
    //TODO    
}

void setupCalc_lj(){
    //TODO
}
__global__ void testCalcLJKernel(){
    //TODO
}

void testGeneratePoints(){
    //init atoms, environment
    int numberOfAtoms = 10;
    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, 0.0, 0.0, 0.0);
    }
    Environment stableEnviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 122.0, numberOfAtoms);

    Environment *enviro = &stableEnviro;

    //calculate size of atoms and environemnt structs
    size_t atomsSize = sizeof(*atoms);
    size_t enviroSize = sizeof(*enviro);

    //declare device structs
    Atom *dev_atoms = new Atom[numberOfAtoms];
    Environment *dev_enviro;

    //allocate memory for device structs
    hipMalloc( (void**) &dev_atoms, atomsSize);
    hipMalloc( (void**) &dev_enviro, enviroSize);

    //copy local structs to device structs on device
    hipMemcpy(dev_atoms, atoms, atomsSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_enviro, enviro, enviroSize, hipMemcpyHostToDevice);

    //allocate memory on device for random number generator state
    hiprandState* devStates;
    hipMalloc ( &devStates, numberOfAtoms*sizeof( hiprandState ) );
                
    // setup seeds
    setup_generator <<<5, 2>>> ( devStates, time(NULL) );

    // generate random numbers
    generatePoints <<<5, 2>>> ( devStates, dev_atoms, dev_enviro );

    //copy atoms back to host
    hipMemcpy(atoms, dev_atoms, atomsSize, hipMemcpyDeviceToHost);

    //assert that all atoms positions are in range of the box
    for (int i = 0; i < numberOfAtoms; i++){
        double dim_x = atoms[i].x;
        double dim_y = atoms[i].y;
        double dim_z = atoms[i].z;

        assert(dim_x >= 0.0 && dim_x <= enviro->x &&
               dim_y >= 0.0 && dim_y <= enviro->y &&
               dim_z >= 0.0 && dim_z <= enviro->z);
    }
    printf("testGeneratePoints successful.");
}

void testCalcEnergy(){
    //TODO    
}

int main(){
    testGeneratePoints();    
    return 0;
}
