#include "hip/hip_runtime.h"
#include "parallelTest.cuh"

void setupGetXFromIndex(){
    int numberOfBlocks = 3;
    int threadsPerBlock = 2;

    int *xValues;
    int *xValues_device;
    
    size_t xSize = numberOfBlocks * threadsPerBlock * sizeof(int);
    xValues = (int *)malloc(xSize);
    hipMalloc((void **) &xValues_device, xSize);
    
    testGetXKernel <<<numberOfBlocks, threadsPerBlock>>>(xValues_device);

    hipMemcpy(xValues, xValues_device, xSize, hipMemcpyDeviceToHost);

    assert(xValues[0] == 1);
    assert(xValues[1] == 2);
    assert(xValues[2] == 2);
    assert(xValues[3] == 3);
    assert(xValues[4] == 3);
    assert(xValues[5] == 3);

    printf("getXFromIndex Correct\n");

    hipFree(xValues_device);
    free(xValues);
}
__global__ void testGetXKernel(int *xValues){
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    
    //xValues[idx] = getXFromIndex(idx); 
}

void setupGetYFromIndex(){
    //TODO
}
__global__ void testGetYKernel(){
    //TODO
}

void setupMakePeriodic(){
    //TODO
}

__global__ void testMakePeriodicKernel(){
    //TODO
}

void setupWrapBox(){
    //TODO
}
__global__ void testWrapBoxKernel(){
    //TODO    
}

void setupCalc_lj(){
    //TODO
}
__global__ void testCalcLJKernel(){
    //TODO
}

void testGeneratePoints(){
    //init atoms, environment
    int numberOfAtoms = 10;
    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, -1.0, -1.0, -1.0);
    }
    Environment stableEnviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 122.0, numberOfAtoms);

    Environment *enviro = &stableEnviro;

    //calculate size of atoms and environemnt structs
    size_t atomsSize = sizeof(*atoms);
    size_t enviroSize = sizeof(*enviro);

    //declare device structs
    Atom *dev_atoms = new Atom[numberOfAtoms];
    Environment *dev_enviro;

    //allocate memory for device structs
    hipMalloc( (void**) &dev_atoms, atomsSize);
    hipMalloc( (void**) &dev_enviro, enviroSize);

    //copy local structs to device structs on device
    hipMemcpy(dev_atoms, atoms, atomsSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_enviro, enviro, enviroSize, hipMemcpyHostToDevice);

    //allocate memory on device for random number generator state
    hiprandState* devStates;
    hipMalloc ( &devStates, numberOfAtoms*sizeof( hiprandState ) );
                
    // setup seeds
    setup_generator <<<5, 2>>> ( devStates, time(NULL) );

    // generate random numbers
    generatePoints <<<5, 2>>> ( devStates, dev_atoms, dev_enviro );

    //copy atoms back to host
    hipMemcpy(atoms, dev_atoms, atomsSize, hipMemcpyDeviceToHost);

    //assert that all atoms positions are in range of the box
    for (int i = 0; i < numberOfAtoms; i++){
        double dim_x = atoms[i].x;
        double dim_y = atoms[i].y;
        double dim_z = atoms[i].z;
        
        printf("%f, %f, %f\n", dim_x, dim_y, dim_z);

        assert(dim_x >= 0.0 && dim_x <= enviro->x &&
               dim_y >= 0.0 && dim_y <= enviro->y &&
               dim_z >= 0.0 && dim_z <= enviro->z);
    }
    printf("testGeneratePoints successful.");
}

void testCalcEnergy(){
        
	struct timeval le_tvBegin, le_tvEnd, pl_tvBegin, pl_tvEnd;

	//Generate enviorment and atoms
	 int numberOfAtoms = 10;
	 Environment enviro = createEnvironment(5.0, 10.0, 15.0, 1.0, 122.0, numberOfAtoms);
	 
    Atom *atoms = new Atom[numberOfAtoms];
    for (int i = 0; i < numberOfAtoms; i++){
        atoms[i] = createAtom(i, rand()*enviro.x, rand()*enviro.y, rand()*enviro.z);
    }
	 
	 //make copies of enviornment and atoms for 
	 //parallel portion
	 Environment enviro2 = enviro;
	 
	 Atom *atoms2 = new Atom[numberOfAtoms];
	 memcpy(atoms2,atoms,numberOfAtoms*sizeof(Atom) );
	 
	 	 
	 /*
	 ** Run the Calculation as Linear.
	 */
	 
	  gettimeofday(&le_tvBegin,NULL); //start clock for execution time
	 
	  double te_linear = calculate_energy(atoms, enviro);
	  
	  gettimeofday(&le_tvEnd,NULL); //start clock for execution time
	  long le_runTime = timevaldiff(&le_tvBegin,&le_tvEnd); //get difference in time in milli seconds

	 	 	 
	 
	 /*
	 ** Run the Calculation as Parallel
	 */
	 
	 gettimeofday(&pl_tvBegin,NULL); //start clock for execution time
	  
	 double te_parallel =  calcEnergyWrapper(atoms2, enviro2);	 
	 
	 gettimeofday(&pl_tvEnd,NULL); //start clock for execution time
	 long pl_runTime = timevaldiff(&pl_tvBegin,&pl_tvEnd); //get difference in time in milli seconds

	 
	 /*
	 ** Print out Results
	 */
	 if( te_parallel == te_linear)
	    printf("testCalcEnergy sucessful\n Both total energies equate to the same value.\n");
	 else
	 	 printf("testCalcEnergy failed\n Both total energies equate to different values.\n");
	 
         printf("Number of elements: %d", numberOfAtoms);
	 printf("Linear Total Energy: %f \n", te_linear);
	 printf("In %d ms", le_runTime);
	 printf("Parallel Total Energy: %f \n", te_parallel);
	 printf("In %d ms", pl_runTime);

    
}

int main(){
    //testGeneratePoints(); 
	 testCalcEnergy();   
    return 0;
}
