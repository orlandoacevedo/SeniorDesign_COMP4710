#include "copyTests.cuh"

void testCopyMolecules(){
    Molecule *molecs;
    int numOfMolec = 10;
    size_t molecSize = sizeof(Molecule) * numOfMolec;
    molecs = (Molecule *)malloc(molecSize);

    for(int i = 0; i < numOfMolec; i++){
        Molecule m = molecs[i];
        
        int atomCount = 3;
        size_t atomSize = sizeof(Atom) * atomCount;
        m.atoms = (Atom *)malloc(atomSize);
        m.atoms[0] = createAtom(1, 1, 1, 1);
        m.atoms[1] = createAtom(2, 1, 1, 1);
        m.atoms[2] = createAtom(3, 1, 2, 3);

        int bondCount = 2;
        size_t bondSize = sizeof(Bond) * bondCount;
        m.bonds = (Bond *)malloc(bondSize);
        m.bonds[0] = createBond(1, 2, 1.2, false);
        m.bonds[1] = createBond(2, 3, 3.1, true);

        int angleCount = 2;
        size_t angleSize = sizeof(Angle) * angleCount;
        m.angles = (Angle *)malloc(angleSize);
        m.angles[0] = createAngle(1, 2, 86, false);
        m.angles[1] = createAngle(1, 3, 180, true);

        int dihedralCount = 2;
        size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
        m.dihedrals = (Dihedral *)malloc(dihedralSize);
        m.dihedrals[0] = createDihedral(1, 2, 65, true);
        m.dihedrals[1] = createDihedral(1, 3, 43, false);

        int hopCount = 2;
        size_t hopSize = sizeof(Hop) * hopCount;
        m.hops = (Hop *)malloc(hopSize);
        m.hops[0] = createHop(1,2,1);
        m.hops[1] = createHop(2,3,1);
        
        m.id = i;
        m.numOfAtoms = atomCount;
        m.numOfBonds = bondCount;
        m.numOfAngles = angleCount;
        m.numOfDihedrals = dihedralCount;
        m.numOfHops = hopCount;

        molecs[i] = m;
    }

    //start cuda-ing
    Molecule *molec_d;
    Molecule molec2;
    printf("Allocating on the device.\n");
    allocateOnDevice(molec_d, molecs, numOfMolec);

    printf("Copying to the device\n");
    moleculeDeepCopyToDevice(molec_d, molecs, numOfMolec);

    /******
      Tests and assert statements.
    ******/

    printf("molec2.id = %d,  before copy\n", molec2.id);
    hipMemcpy(&molec2, molec_d, sizeof(Molecule), hipMemcpyDeviceToHost); 
    //moleculeDeepCopyToHost(&molec2, molec_d);

    printf("molec.id = %d, molec2.id = %d\n", molecs[0].id, molec2.id);
    printf("molec.numOfAtoms = %d, molec2.numOfAtoms = %d\n", molecs[0].numOfAtoms, molec2.numOfAtoms);
    printf("molec.numOfBonds = %d, molec2.numOfBonds = %d\n", molecs[0].numOfBonds, molec2.numOfBonds);
    printf("molec.numOfAngles = %d, molec2.numOfAngles = %d\n", molecs[0].numOfAngles, molec2.numOfAngles);
    printf("molec.numOfDihedrals = %d, molec2.numOfDihedrals = %d\n", molecs[0].numOfDihedrals, molec2.numOfDihedrals);
    printf("molec.numOfHops = %d, molec2.numOfHops = %d\n", molecs[0].numOfHops, molec2.numOfHops);
    
    
    /*assert(molec.id == molec2.id);
    assert(molec.numOfAtoms == molec2.numOfAtoms);
    assert(molec.numOfBonds == molec2.numOfBonds);
    assert(molec.numOfAngles == molec2.numOfAngles);
    assert(molec.numOfDihedrals == molec2.numOfDihedrals);
    assert(molec.numOfHops == molec2.numOfHops);*/
}

void testAllocateMemory(){
    int numOfMolecules = 3;

    Molecule *molec;
    Molecule *molec_d;
    size_t molecSize = sizeof(Molecule) * numOfMolecules;
    molec = (Molecule *)malloc(molecSize);

    for(int i = 0; i < numOfMolecules; i++){
        printf("Creating %dth array.\n", i);
        Atom *atoms;
        int atomCount = 3;
        size_t atomSize = sizeof(Atom) * atomCount;
        atoms = (Atom *)malloc(atomSize);
        atoms[0] = createAtom(1, 1, 1, 1);
        atoms[1] = createAtom(2, 1, 1, 1);
        atoms[2] = createAtom(3, 1, 2, 3);

        Bond *bonds;
        int bondCount = 2;
        size_t bondSize = sizeof(Bond) * bondCount;
        bonds = (Bond *)malloc(bondSize);
        bonds[0] = createBond(1, 2, 1.2, false);
        bonds[1] = createBond(2, 3, 3.1, true);

        Angle *angles;
        int angleCount = 2;
        size_t angleSize = sizeof(Angle) * angleCount;
        angles = (Angle *)malloc(angleSize);
        angles[0] = createAngle(1, 2, 86, false);
        angles[1] = createAngle(1, 3, 180, true);


        Dihedral *dihedrals;
        int dihedralCount = 2;
        size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
        dihedrals = (Dihedral *)malloc(dihedralSize);
        dihedrals[0] = createDihedral(1, 2, 65, true);
        dihedrals[1] = createDihedral(1, 3, 43, false);

        Hop *hops;
        int hopCount = 2;
        size_t hopSize = sizeof(Hop) * hopCount;
        hops = (Hop *)malloc(hopSize);
        hops[0] = createHop(1,2,1);
        hops[1] = createHop(2,3,1);
        
        molec[i] = createMolecule(i + 1,
                atoms, angles, bonds, dihedrals, hops,
                atomCount, angleCount, bondCount, dihedralCount, hopCount);

        free(atoms);
        free(angles);
        free(bonds);
        free(dihedrals);
        free(hops);
    }
    
    allocateOnDevice(molec_d, molec, numOfMolecules);
    printf("Allocated on Device\n");
}

void testFreeMemory(){
    //TODO
}
