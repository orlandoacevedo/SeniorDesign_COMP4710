#include "copyTests.cuh"

void testCopyMolecules(){
    //Molecules on host
    Molecule *molecs;
    Molecule *copiedMolecs;
    
    int numOfMolecules = 3;
    size_t molecSize = sizeof(Molecule) * numOfMolecules;
    molecs = (Molecule *)malloc(molecSize);
    copiedMolecs = (Molecule *)malloc(molecSize);

    int angleCount = 2;
    int dihedralCount = 2;
    int bondCount = 2;
    int atomCount = 3;
    int hopCount = 2;

    for(int i = 0; i < numOfMolecules; i++){
        printf("Creating molecule %d\n.", i);
        
        Molecule m = molecs[i];
        
        size_t atomSize = sizeof(Atom) * atomCount;
        copiedMolecs[i].atoms = (Atom *)malloc(atomSize);

        m.atoms = (Atom *)malloc(atomSize);
        m.atoms[0] = createAtom(1, 1, 1, 1);
        m.atoms[1] = createAtom(2, 2, 2, 2);
        m.atoms[2] = createAtom(3, 3, 3, 3);

        size_t bondSize = sizeof(Bond) * bondCount;
        copiedMolecs[i].bonds = (Bond *)malloc(bondSize);

        m.bonds = (Bond *)malloc(bondSize);
        m.bonds[0] = createBond(1, 2, 1.2, false);
        m.bonds[1] = createBond(2, 3, 3.1, true);

        size_t angleSize = sizeof(Angle) * angleCount;
        copiedMolecs[i].angles = (Angle *)malloc(angleSize);

        m.angles = (Angle *)malloc(angleSize);
        m.angles[0] = createAngle(1, 2, 86, false);
        m.angles[1] = createAngle(1, 3, 180, true);

        size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
        copiedMolecs[i].dihedrals = (Dihedral *)malloc(dihedralSize);

        m.dihedrals = (Dihedral *)malloc(dihedralSize);
        m.dihedrals[0] = createDihedral(1, 2, 65, true);
        m.dihedrals[1] = createDihedral(1, 3, 43, false);

        size_t hopSize = sizeof(Hop) * hopCount;
        copiedMolecs[i].hops = (Hop *)malloc(hopSize);
        m.hops = (Hop *)malloc(hopSize);
        
        m.hops[0] = createHop(1,3,2);
        m.hops[1] = createHop(2,3,1);
        
        m.id = i * atomCount + 57;
        m.numOfAtoms = atomCount;
        m.numOfBonds = bondCount;
        m.numOfAngles = angleCount;
        m.numOfDihedrals = dihedralCount;
        m.numOfHops = hopCount;

        molecs[i] = m;
    }

    printf("Testing deep copy to device.\n");
    size_t atomSize = numOfMolecules * atomCount * sizeof(Atom);
    size_t bondSize = numOfMolecules * bondCount * sizeof(Bond);
    size_t angleSize = numOfMolecules * angleCount * sizeof(Angle);
    size_t dihedralSize = numOfMolecules * dihedralCount * sizeof(Dihedral);
    size_t hopSize = numOfMolecules * hopCount * sizeof(Hop);
    size_t deviceMolecSize = numOfMolecules * sizeof(DeviceMolecule);

    Atom *atoms_d;
    Bond *bonds_d;
    Angle *angles_d;
    Dihedral *dihedrals_d;
    Hop *hops_d;
    DeviceMolecule *molec_d;
 
    hipMalloc((void **) &molec_d, deviceMolecSize);
    hipMalloc((void **) &atoms_d, atomSize);
    hipMalloc((void **) &bonds_d, bondSize);
    hipMalloc((void **) &angles_d, angleSize);
    hipMalloc((void **) &dihedrals_d, dihedralSize);
    hipMalloc((void **) &hops_d, hopSize);
    
    moleculeDeepCopyToDevice(molec_d, molecs, numOfMolecules, atoms_d,
            bonds_d, angles_d, dihedrals_d, hops_d);

    moleculeDeepCopyToHost(copiedMolecs, molec_d, numOfMolecules, atoms_d, bonds_d,
            angles_d, dihedrals_d, hops_d);

    printf("Testing molecules.\n");
    for(int i = 0; i < numOfMolecules; i++){
        Molecule m = molecs[i];
        Molecule dm = copiedMolecs[i];
        
        assert(dm.id == m.id);
        assert(dm.numOfAtoms == m.numOfAtoms);
        assert(dm.numOfBonds == m.numOfBonds);
        assert(dm.numOfAngles == m.numOfAngles);
        assert(dm.numOfDihedrals == m.numOfDihedrals);
        assert(dm.numOfHops == m.numOfHops);
        
        for(int j = 0; j < copiedMolecs[i].numOfAtoms; j++){
            Atom a1 = copiedMolecs[i].atoms[j];
            Atom a2 = molecs[i].atoms[j];
            assert(a1.x == a2.x);
            assert(a1.y == a2.y);
            assert(a1.z == a2.z);
            assert(a1.id == a2.id);
            assert(a1.sigma == a2.sigma);
            assert(a1.epsilon == a2.epsilon);
            assert(a1.charge == a2.charge);
        }
        for(int j = 0; j < copiedMolecs[i].numOfBonds; j++){
            Bond a1 = copiedMolecs[i].bonds[j];
            Bond a2 = molecs[i].bonds[j];
            assert(a1.atom1 == a2.atom1);
            assert(a1.atom2 == a2.atom2);
            assert(a1.distance == a2.distance);
            assert(a1.variable == a2.variable);
        }
        for(int j = 0; j < copiedMolecs[i].numOfAngles; j++){
            Angle a1 = copiedMolecs[i].angles[j];
            Angle a2 = molecs[i].angles[j];
            assert(a1.atom1 == a2.atom1);
            assert(a1.atom2 == a2.atom2);
            assert(a1.value == a2.value);
            assert(a1.variable == a2.variable);
        }
        for(int j = 0; j < copiedMolecs[i].numOfDihedrals; j++){
            Dihedral a1 = copiedMolecs[i].dihedrals[j];
            Dihedral a2 = molecs[i].dihedrals[j];
            assert(a1.atom1 == a2.atom1);
            assert(a1.atom2 == a2.atom2);
            assert(a1.value == a2.value);
            assert(a1.variable == a2.variable);
        }
        for(int j = 0; j < copiedMolecs[i].numOfHops; j++){
            Hop a1 = copiedMolecs[i].hops[j];
            Hop a2 = molecs[i].hops[j];
            assert(a1.atom1 == a2.atom1);
            assert(a1.atom2 == a2.atom2);
            assert(a1.hop == a2.hop);
        }

    }
    printf("testCopyMolecules completed successfully.\n");
}
