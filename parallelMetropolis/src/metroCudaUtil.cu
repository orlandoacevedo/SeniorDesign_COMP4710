#include "hip/hip_runtime.h"
/*!\file*/
#include "metroCudaUtil.cuh"

void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
    if( hipSuccess != err) {                                                
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                file, line, hipGetErrorString(err) );
    } 
}

DeviceMolecule createDeviceMolecule(int id, int atomStart, int numOfAtoms,
        int bondStart, int numOfBonds, int angleStart, int numOfAngles,
        int dihedralStart, int numOfDihedrals, int hopStart, int numOfHops){
    
    DeviceMolecule dm;
    dm.id = id;
    
    dm.atomStart = atomStart;
    dm.numOfAtoms = numOfAtoms;
    
    dm.bondStart = bondStart;
    dm.numOfBonds = numOfBonds;

    dm.angleStart = angleStart;
    dm.numOfAngles = numOfAngles;

    dm.dihedralStart = dihedralStart;
    dm.numOfDihedrals = numOfDihedrals;

    dm.hopStart = hopStart;
    dm.numOfHops = numOfHops;

    return dm;
}

__device__ int getXFromIndex(int idx){
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

__device__ int getYFromIndex(int x, int idx){
    return idx - (x * x - x) / 2;
}

__device__ double makePeriodic(double x, double box){
    while(x < -0.5 * box){
        x += box;
    }

    while(x > 0.5 * box){
        x -= box;
    }

    return x;
}

double wrapBox(double x, double box){
    while(x >  box){
        x -= box;
    }
    while(x < 0){
        x += box;
    }

    return x;
}

void keepMoleculeInBox(Molecule *molecule, Environment *enviro){
    double maxX = DBL_MIN;
    double maxY = DBL_MIN;
    double maxZ = DBL_MIN;

    double minX = DBL_MAX;
    double minY = DBL_MAX;
    double minZ = DBL_MAX;

    double nudge = pow(10.0, -15.0);

    //determine extreme boundaries for molecule
    for (int i = 0; i < molecule->numOfAtoms; i++){
        double currentX = molecule->atoms[i].x;
        double currentY = molecule->atoms[i].y;
        double currentZ = molecule->atoms[i].z;

        if (currentX > maxX)
           maxX = currentX;
        else if (currentX < minX)
           minX = currentX;

        if (currentY > maxY)
            maxY = currentY;
        else if (currentY < minY)
            minY = currentY;

        if (currentZ > maxZ)
            maxZ = currentZ;
        else if (currentZ < minZ)
            minZ = currentZ;
    
    }

    bool isFullyOutX = (minX > enviro->x || maxX < 0) ? true : false;
    bool isFullyOutY = (minY > enviro->y || maxY < 0) ? true : false;
    bool isFullyOutZ = (minZ > enviro->z || maxZ < 0) ? true : false;

    //for each axis, determine if the molecule escapes the environment 
    //and wrap it around into the environment
    for (int i = 0; i < molecule->numOfAtoms; i++){
        double* currentX = &(molecule->atoms[i].x);
        double* currentY = &(molecule->atoms[i].y);
        double* currentZ = &(molecule->atoms[i].z);
        if (maxX > enviro->x){
            if (!isFullyOutX){
                *currentX += (enviro->x - minX);
            }
            *currentX = wrapBox(*currentX + nudge, enviro->x);
        }
        else if (minX < 0){
            if (!isFullyOutX)
                *currentX -= maxX;
            *currentX = wrapBox(*currentX - nudge, enviro->x);
        }

        if (maxY > enviro->y){
            if (!isFullyOutY)
                *currentY += (enviro->y - minY);
            *currentY = wrapBox(*currentY + nudge, enviro->y);
        }
        else if (minY < 0){
            if (!isFullyOutY)
                *currentY -= maxY;
            *currentY = wrapBox(*currentY - nudge, enviro->y);
        }

        if (maxZ > enviro->z){
            if (!isFullyOutZ)
                *currentZ += (enviro->z - minZ);
            *currentZ = wrapBox(*currentZ + nudge, enviro->z);
        }
        else if (minZ < 0){
            if (!isFullyOutZ)
                *currentZ -= maxZ;
            *currentZ = wrapBox(*currentZ - nudge, enviro->z);
        }
    }
}

__device__ double calc_lj(Atom atom1, Atom atom2, Environment enviro){
    //store LJ constants locally
    double sigma = calcBlending(atom1.sigma, atom2.sigma);
    double epsilon = calcBlending(atom1.epsilon, atom2.epsilon);
    
    //calculate difference in coordinates
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;

    //calculate distance between atoms
    deltaX = makePeriodic(deltaX, enviro.x);
    deltaY = makePeriodic(deltaY, enviro.y);
    deltaZ = makePeriodic(deltaZ, enviro.z);

    const double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);

    //calculate terms
    const double sig2OverR2 = pow(sigma, 2) / r2;
    const double sig6OverR6 = pow(sig2OverR2, 3);
    const double sig12OverR12 = pow(sig6OverR6, 2);
    const double energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
    
    if (r2 == 0){
        return 0.0;
    }
    else{
        return energy;
    }
}

__global__ void assignAtomPositions(double *dev_doublesX, double *dev_doublesY, double *dev_doublesZ, Atom *atoms, Environment *enviro){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    //for each atom...
    if (idx < enviro->numOfAtoms){
        atoms[idx].x = dev_doublesX[idx] * enviro->x + atoms[idx].x;
        atoms[idx].y = dev_doublesY[idx] * enviro->y + atoms[idx].y;
        atoms[idx].z = dev_doublesZ[idx] * enviro->z + atoms[idx].z;
    }
}

void generatePoints(Atom *atoms, Environment *enviro){
    //setup CUDA storage
    hiprandGenerator_t generator;
    double *devXDoubles;
    double *devYDoubles;
    double *devZDoubles;
    //double *hostDoubles;
    Atom *devAtoms;
    Environment *devEnviro;

    //allocate memory on device
    hipMalloc((void**)&devXDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devYDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devZDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devAtoms, enviro->numOfAtoms * sizeof(Atom));
    hipMalloc((void**)&devEnviro, sizeof(Environment));

    //copy local data to device
    hipMemcpy(devAtoms, atoms, enviro->numOfAtoms * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(devEnviro, enviro, sizeof(Environment), hipMemcpyHostToDevice);

    //generate doubles for all coordinates
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned int) time(NULL));
    hiprandGenerateUniformDouble(generator, devXDoubles, enviro->numOfAtoms);
    hiprandGenerateUniformDouble(generator, devYDoubles, enviro->numOfAtoms);
    hiprandGenerateUniformDouble(generator, devZDoubles, enviro->numOfAtoms);

    //calculate number of blocks required
    int numOfBlocks = enviro->numOfAtoms / THREADS_PER_BLOCK + (enviro->numOfAtoms % THREADS_PER_BLOCK == 0 ? 0 : 1);

    //assign the doubles to the coordinates
    assignAtomPositions <<< numOfBlocks, THREADS_PER_BLOCK >>> (devXDoubles, devYDoubles, devZDoubles, devAtoms, devEnviro);

    //copy the atoms back to host
    hipMemcpy(atoms, devAtoms, enviro->numOfAtoms * sizeof(Atom), hipMemcpyDeviceToHost);

    //cleanup
    hiprandDestroyGenerator(generator);
    hipFree(devXDoubles);
    hipFree(devYDoubles);
    hipFree(devZDoubles);
    hipFree(devAtoms);
    hipFree(devEnviro);
}

void generatePoints(Molecule *molecules, Environment *enviro){
    srand(time(NULL));

    for (int i = 0; i < enviro->numOfMolecules; i++){
        double baseX = ( (double) rand() / RAND_MAX) * enviro->x;
        double baseY = ( (double) rand() / RAND_MAX) * enviro->y;
        double baseZ = ( (double) rand() / RAND_MAX) * enviro->z;
        for (int j = 0; j < molecules[i].numOfAtoms; j++){
            molecules[i].atoms[j].x += baseX;
            molecules[i].atoms[j].y += baseY;
            molecules[i].atoms[j].z += baseZ;
        }

        keepMoleculeInBox(&(molecules[i]), enviro);
    }
}

double calcEnergyWrapper(Molecule *molecules, Environment *enviro){
    
    Atom *atoms = (Atom *) malloc(sizeof(Atom) * enviro->numOfAtoms);
    int atomIndex = 0;
    for(int i = 0; i < enviro->numOfMolecules; i++){
        Molecule currentMolecule = molecules[i];
        for(int j = 0; j < currentMolecule.numOfAtoms; j++){
            atoms[atomIndex] = currentMolecule.atoms[j];
            atomIndex++;
        }
    }

    return calcEnergyWrapper(atoms, enviro, molecules);
}

double calcEnergyWrapper(Atom *atoms, Environment *enviro, Molecule *molecules){
    //setup CUDA storage
    double totalEnergy = 0.0;
    Atom *atoms_device;
    double *energySum_device;
    double *energySum_host;
    Environment *enviro_device;

    //calculate CUDA thread mgmt
    int N =(int) ( pow( (float) enviro->numOfAtoms,2)-enviro->numOfAtoms)/2;
    int blocks = N / THREADS_PER_BLOCK + (N % THREADS_PER_BLOCK == 0 ? 0 : 1); 

    //The number of bytes of shared memory per block of
    //size_t sharedSize = sizeof(double) * THREADS_PER_BLOCK;
    
    size_t atomSize = enviro->numOfAtoms * sizeof(Atom);
    size_t energySumSize = N * sizeof(double);
    
    //allocate memory on the device
    energySum_host = (double *) malloc(energySumSize);
    hipMalloc((void **) &atoms_device, atomSize);
    hipMalloc((void **) &energySum_device, energySumSize);
    hipMalloc((void **) &enviro_device, sizeof(Environment));

    //copy data to the device
    cudaErrorCheck(hipMemcpy(atoms_device, atoms, atomSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(enviro_device, enviro, sizeof(Environment), hipMemcpyHostToDevice));

    if (molecules != NULL){
        int bondCount = 0;
        int angleCount = 0;
        int dihedralCount = 0;
        int hopCount = 0;
        for (int i = 0; i < enviro->numOfMolecules; i++){
            bondCount += molecules[i].numOfBonds;
            angleCount += molecules[i].numOfAngles;
            dihedralCount += molecules[i].numOfDihedrals;
            hopCount += molecules[i].numOfHops;
        }

        size_t dMolecSize = sizeof(DeviceMolecule) * enviro->numOfMolecules;
        size_t bondSize = sizeof(Bond) * bondCount;
        size_t angleSize = sizeof(Angle) * angleCount;
        size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
        size_t hopSize = sizeof(Hop) * hopCount;
        
        DeviceMolecule *molec_d;
        Bond *bonds_d;
        Angle *angles_d;
        Dihedral *dihedrals_d;
        Hop *hops_d;
        
        hipMalloc((void **) &molec_d, dMolecSize);
        hipMalloc((void **) &bonds_d, bondSize);
        hipMalloc((void **) &angles_d, angleSize);
        hipMalloc((void **) &dihedrals_d, dihedralSize);
        hipMalloc((void **) &hops_d, hopSize);


        moleculeDeepCopyToDevice(molec_d, molecules, enviro->numOfMolecules, atoms_device, bonds_d, angles_d, dihedrals_d, hops_d);

        calcEnergy <<<blocks, THREADS_PER_BLOCK>>>(atoms_device, enviro_device, energySum_device, molec_d, hops_d);

    hipFree(molec_d);
    hipFree(bonds_d);
    hipFree(angles_d);
    hipFree(dihedrals_d);
    hipFree(hops_d);    
    }
    else{
        calcEnergy <<<blocks, THREADS_PER_BLOCK>>>(atoms_device, enviro_device, energySum_device);
    }
    
    cudaErrorCheck(hipMemcpy(energySum_host, energySum_device, energySumSize, hipMemcpyDeviceToHost));

    for(int i = 0; i < N; i++){

        //get atom IDs for each calculation
        int c = -2 * i;
        int discriminant = 1 - 4 * c;
        int qv = (-1 + sqrtf(discriminant)) / 2;
        int atomXid = qv + 1;
        
        int atomYid =  i - (atomXid * atomXid - atomXid) / 2;
        
        //check for stray calculations that returned invalid results
        if (isnan(energySum_host[i]) != 0 || isinf(energySum_host[i]) != 0){
            energySum_host[i] = calcEnergyOnHost(atoms[atomXid], atoms[atomYid], enviro, molecules);
        }
           
        //sum up energies 
        totalEnergy += energySum_host[i];
    }

    //cleanup
    hipFree(atoms_device);
    hipFree(energySum_device);
    free(energySum_host);

    return totalEnergy;
}

double calcEnergyOnHost(Atom atom1, Atom atom2, Environment *enviro, Molecule *molecules){
    //define terms
    const double e = 332.06;
    double sigma = sqrt(atom1.sigma * atom2.sigma);
    double epsilon = sqrt(atom1.epsilon * atom2.epsilon);
    
    //calculate distance between atoms
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;
  
    deltaX = make_periodic(deltaX, enviro->x);
    deltaY = make_periodic(deltaY, enviro->y);
    deltaZ = make_periodic(deltaZ, enviro->z);

    double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);

    double r = sqrt(r2);

    //combine terms and calculate energies
    double sig2OverR2 = pow(sigma, 2) / r2;
    double sig6OverR6 = pow(sig2OverR2, 3);
    double sig12OverR12 = pow(sig6OverR6, 2);
    double lj_energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);

    double charge_energy = (atom2.charge * atom1.charge * e) / r;
    
    //check if atoms overlap
    if (r2 == 0.0){
        lj_energy = 0.0;
        charge_energy = 0.0;
    }

    double fValue = 1.0;

    if (molecules != NULL)
        fValue = getFValueHost(atom1, atom2, molecules, enviro);

    return fValue * (lj_energy + charge_energy);

}

__global__ void calcEnergy(Atom *atoms, Environment *enviro, double *energySum, DeviceMolecule *dev_molecules, Hop *hops){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double lj_energy, charge_energy, fValue;

    int N =(int) ( pow( (float) enviro->numOfAtoms,2)-enviro->numOfAtoms)/2;

    if(idx < N ){
        //calculate the x and y positions in the Atom array
        int xAtom_pos, yAtom_pos;
        xAtom_pos = getXFromIndex(idx);
        yAtom_pos = getYFromIndex(xAtom_pos, idx);

        Atom xAtom, yAtom;
        xAtom = atoms[xAtom_pos];
        yAtom = atoms[yAtom_pos];

        if(xAtom.sigma < 0 || xAtom.epsilon < 0 || yAtom.sigma < 0 || yAtom.epsilon < 0){
            energySum[idx] = 0.0;
        }
        else{
            lj_energy = calc_lj(xAtom,yAtom,*enviro);
            charge_energy = calcCharge(xAtom, yAtom, enviro);
            double fValue = 1.0;
            if (dev_molecules != NULL){
               fValue = getFValue(xAtom, yAtom, dev_molecules, enviro, hops);
            }
            
            energySum[idx] = fValue * (lj_energy + charge_energy);
        }
    }
}

__device__ double calcCharge(Atom atom1, Atom atom2, Environment *enviro){
    const double e = 332.06;
 
    //calculate difference in coordinates
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;

    //calculate distance between atoms
    deltaX = makePeriodic(deltaX, enviro->x);
    deltaY = makePeriodic(deltaY, enviro->y);
    deltaZ = makePeriodic(deltaZ, enviro->z);

    double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);
    
    double r = sqrt(r2);


    if (r == 0.0){
        return 0.0;
    }
    else{
        return (atom1.charge * atom2.charge * e) / r;
    }
}

__device__ double calcBlending(double d1, double d2){
    return sqrt(d1 * d2);
}

__device__ int getMoleculeFromAtomID(Atom a1, DeviceMolecule *dev_molecules, Environment enviro){
    int atomId = a1.id;
    int currentIndex = enviro.numOfMolecules - 1;
    int molecId = dev_molecules[currentIndex].id;
    while(atomId < molecId && currentIndex > 0){
        currentIndex -= 1;
        molecId = dev_molecules[currentIndex].id;
    }
    return molecId;

}

__device__ double getFValue(Atom atom1, Atom atom2, DeviceMolecule *dev_molecules, Environment *enviro, Hop *hops){
    int m1 = getMoleculeFromAtomID(atom1, dev_molecules, *enviro);
    int m2 = getMoleculeFromAtomID(atom2, dev_molecules, *enviro);
    if(m1 != m2){
        return 1.0;
    }
    else{
        int moleculeIndex = 0;
        for (int i = 0; i < enviro->numOfMolecules; i++){
            if (dev_molecules[i].id == m1)
                moleculeIndex = i;
        }
        size_t molecHopSize = sizeof(Hop) * dev_molecules[moleculeIndex].numOfHops;
        Hop *molecHops = (Hop *)malloc(molecHopSize);
        int hopStart = dev_molecules[moleculeIndex].hopStart;
        for (int i = 0; i < dev_molecules[moleculeIndex].numOfHops; i++){
            molecHops[i] = hops[hopStart + i];
        }
        int hopChain = hopGE3(atom1.id, atom2.id, dev_molecules[moleculeIndex], molecHops);
        free(molecHops);
        if (hopChain == 3)
            return 0.5;
        else if (hopChain > 3)
            return 1.0;
        else
            return 0.0;
    } 
}

__device__ int hopGE3(int atom1, int atom2, DeviceMolecule dev_molecule, Hop *molecule_hops){
    for(int x=0; x< dev_molecule.numOfHops; x++){
	    Hop myHop = molecule_hops[x];
	    if((myHop.atom1==atom1 && myHop.atom2==atom2) || (myHop.atom1==atom2 && myHop.atom2==atom1))
	        return myHop.hop;
	 }
	 return 0;
}

Molecule* getMoleculeFromAtomIDHost(Atom a1, Molecule *molecules, Environment enviro){
    int atomId = a1.id;
    int currentIndex = enviro.numOfMolecules - 1;
    int molecId = molecules[currentIndex].id;
    while(atomId < molecId && currentIndex > 0){
        currentIndex -= 1;
        molecId = molecules[currentIndex].id;
    }
    return &molecules[currentIndex];

}

double getFValueHost(Atom atom1, Atom atom2, Molecule *molecules, Environment *enviro){
    Molecule *m1 = getMoleculeFromAtomIDHost(atom1, molecules, *enviro);
    Molecule *m2 = getMoleculeFromAtomIDHost(atom2, molecules, *enviro);
    Molecule molec = molecules[0];
    for(int i = 0; i < enviro->numOfMolecules; i++){
        if(molecules[i].id == m1->id){
            molec = molecules[i];
            break;
        }
    }

    if(m1->id != m2->id)
        return 1.0;
	else{
        int hops = hopGE3Host(atom1.id, atom2.id, *m1);
        if (hops == 3)
            return 0.5;
        else if (hops > 3)
            return 1.0;
        else
            return 0.0;
     }
}

int hopGE3Host(int atom1, int atom2, Molecule molecule){
    for(int x=0; x< molecule.numOfHops; x++){
		      Hop myHop = molecule.hops[x];
				if((myHop.atom1==atom1 && myHop.atom2==atom2) ||
                        (myHop.atom1 == atom2 && myHop.atom2 == atom1) )
				    return myHop.hop;
	 }
	 return 0;
}

void rotateMolecule(Molecule molecule, Atom pivotAtom, double maxRotation){
    //save pivot atom coordinates because they will change
    double pivotAtomX = pivotAtom.x;
    double pivotAtomY = pivotAtom.y;
    double pivotAtomZ = pivotAtom.z;

    //translate entire molecule to place pivotAtom at origin
    for (int i = 0; i < molecule.numOfAtoms; i++){
        molecule.atoms[i].x -= pivotAtomX;
        molecule.atoms[i].y -= pivotAtomY;
        molecule.atoms[i].z -= pivotAtomZ;
    }

    srand(time(NULL));
    double dtr = PI / 180.0;

    //rotate molecule about origin
    for (int axis = 0; axis < 3; axis++){
        double rotation = ((double) rand() / (double) RAND_MAX) * maxRotation * dtr;
        double sinrot = sin(rotation);
        double cosrot = cos(rotation);
        if (axis == 0){ //rotate about x-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldY = thisAtom->y;
                double oldZ = thisAtom->z;
                thisAtom->y = cosrot * oldY + sinrot * oldZ;
                thisAtom->z = cosrot * oldZ - sinrot * oldY;
            }
        }
        else if (axis == 1){ //rotate about y-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldX = thisAtom->x;
                double oldZ = thisAtom->z;
                thisAtom->x = cosrot * oldX - sinrot * oldZ;
                thisAtom->z = cosrot * oldZ + sinrot * oldX;
            }
        }
        if (axis == 2){ //rotate about z-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldX = thisAtom->x;
                double oldY = thisAtom->y;
                thisAtom->x = cosrot * oldX + sinrot * oldY;
                thisAtom->y = cosrot * oldY - sinrot * oldX;
            }
        }
    }

    //translate entire molecule back based on original pivot point
    for (int i = 0; i < molecule.numOfAtoms; i++){
        molecule.atoms[i].x += pivotAtomX;
        molecule.atoms[i].y += pivotAtomY;
        molecule.atoms[i].z += pivotAtomZ;
    }
}

void moleculeDeepCopyToDevice(DeviceMolecule *molec_d, Molecule *molec_h,
        int numOfMolecules, Atom *atoms_d, Bond *bonds_d, Angle *angles_d,
        Dihedral *dihedrals_d, Hop *hops_d){
    
    int atomCount = 0;
    int bondCount = 0;
    int angleCount = 0;
    int dihedralCount = 0;
    int hopCount = 0;

    for(int i = 0; i < numOfMolecules; i++){
        Molecule m = molec_h[i];
        
        atomCount += m.numOfAtoms;
        bondCount += m.numOfBonds;
        angleCount += m.numOfAngles;
        dihedralCount += m.numOfDihedrals;
        hopCount += m.numOfHops;
    }
    
    //size of each array
    size_t molecSize = sizeof(DeviceMolecule) * numOfMolecules;
    size_t atomSize = sizeof(Atom) * atomCount;
    size_t bondSize = sizeof(Bond) * bondCount;
    size_t angleSize = sizeof(Angle) * angleCount;
    size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
    size_t hopSize = sizeof(Hop) * hopCount;
   
    //create arrays to hold data on host
    DeviceMolecule *dMolec_h = (DeviceMolecule *)malloc(molecSize);
    Atom *atoms_h = (Atom *)malloc(atomSize);
    Bond *bonds_h = (Bond *)malloc(bondSize);
    Angle *angles_h = (Angle *)malloc(angleSize);
    Dihedral *dihedrals_h = (Dihedral *)malloc(dihedralSize);
    Hop *hops_h = (Hop *)malloc(hopSize);

    int atomIndex = 0;
    int bondIndex = 0;
    int angleIndex = 0;
    int dihedralIndex = 0;
    int hopIndex = 0;

    //split fields into their own arrays
    for(int i = 0; i < numOfMolecules; i++){
        Molecule m = molec_h[i];
        //Create device molecule
        dMolec_h[i] = createDeviceMolecule(m.id, atomIndex, m.numOfAtoms,
                bondIndex, m.numOfBonds, angleIndex, m.numOfAngles,
                dihedralIndex, m.numOfDihedrals, hopIndex, m.numOfHops);
        
        //assign atoms
        for(int j = 0; j < m.numOfAtoms; j++){
            atoms_h[atomIndex] = m.atoms[j];
            atomIndex++;
        }

        //assign bonds
        for(int j = 0; j < m.numOfBonds; j++){
            bonds_h[bondIndex] = m.bonds[j];
            bondIndex++;
        }
        
        //assign angles
        for(int j = 0; j < m.numOfAngles; j++){
            angles_h[angleIndex] = m.angles[j];
            angleIndex++;
        }
        
        //assign dihedrals
        for(int j = 0; j < m.numOfDihedrals; j++){
            dihedrals_h[dihedralIndex] = m.dihedrals[j];
            dihedralIndex++;
        }

        //assing hops
        for(int j = 0; j < m.numOfHops; j++){
            hops_h[hopIndex] = m.hops[j];
            hopIndex++;
        }
    }

    //transfer data
    cudaErrorCheck(hipMemcpy(molec_d, dMolec_h, molecSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(atoms_d, atoms_h, atomSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(bonds_d, bonds_h, bondSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(angles_d, angles_h, angleSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(dihedrals_d, dihedrals_h, dihedralSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(hops_d, hops_h, hopSize, hipMemcpyHostToDevice));

    free(dMolec_h);
    free(atoms_h);
    free(bonds_h);
    free(angles_h);
    free(dihedrals_h);
    free(hops_h);
}

void moleculeDeepCopyToHost(Molecule *molec_h, DeviceMolecule *molec_d,
        int numOfMolecules,Atom *atoms_d, Bond *bonds_d, Angle *angles_d,
        Dihedral *dihedrals_d, Hop *hops_d){
   
    size_t molecSize = sizeof(DeviceMolecule) * numOfMolecules;
    DeviceMolecule *dMolec_h = (DeviceMolecule *)malloc(molecSize);
    cudaErrorCheck(hipMemcpy(dMolec_h, molec_d, molecSize, hipMemcpyDeviceToHost));

    int atomCount = 0;
    int bondCount = 0;
    int angleCount = 0;
    int dihedralCount = 0;
    int hopCount = 0;
    
    for(int i = 0; i < numOfMolecules; i++){
        DeviceMolecule m = dMolec_h[i];

        //assign correct fields
        molec_h[i].id = m.id;
        molec_h[i].numOfAtoms = m.numOfAtoms;
        molec_h[i].numOfBonds = m.numOfBonds;
        molec_h[i].numOfAngles = m.numOfAngles;
        molec_h[i].numOfDihedrals = m.numOfDihedrals;
        molec_h[i].numOfHops = m.numOfHops;

        atomCount += m.numOfAtoms;
        bondCount += m.numOfBonds;
        angleCount += m.numOfAngles;
        dihedralCount += m.numOfDihedrals;
        hopCount += m.numOfHops;
    }
    size_t atomSize = sizeof(Atom) * atomCount;
    size_t bondSize = sizeof(Bond) * bondCount;
    size_t angleSize = sizeof(Angle) * angleCount;
    size_t dihedralSize = sizeof(Dihedral) * dihedralCount;
    size_t hopSize = sizeof(Hop) * hopCount;
    
    Atom *atoms_h = (Atom *)malloc(atomSize);
    Bond *bonds_h = (Bond *)malloc(bondSize);
    Angle *angles_h = (Angle *)malloc(angleSize);
    Dihedral *dihedrals_h = (Dihedral *)malloc(dihedralSize);
    Hop *hops_h = (Hop *)malloc(hopSize);

    cudaErrorCheck(hipMemcpy(atoms_h, atoms_d, atomSize, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(bonds_h, bonds_d, bondSize, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(angles_h, angles_d, angleSize, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(dihedrals_h, dihedrals_d, dihedralSize, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(hops_h, hops_d, hopSize, hipMemcpyDeviceToHost));


    for(int i = 0; i < numOfMolecules; i++){
        DeviceMolecule dm = dMolec_h[i];
        
        Molecule m = molec_h[i];
        //atoms
        for(int j = 0; j < m.numOfAtoms; j++){
            molec_h[i].atoms[j] = atoms_h[j + dm.atomStart];
        }

        //bonds
        for(int j = 0; j < m.numOfBonds; j++){
            molec_h[i].bonds[j] = bonds_h[j + dm.bondStart];
        }
        //angles
        for(int j = 0; j < m.numOfAngles; j++){
            molec_h[i].angles[j] = angles_h[j + dm.angleStart];
        }
        //dihedrals
        for(int j = 0; j < m.numOfDihedrals; j++){
            molec_h[i].dihedrals[j] = dihedrals_h[j + dm.dihedralStart];
        }
        //hops
        for(int j = 0; j < m.numOfHops; j++){
            molec_h[i].hops[j] = hops_h[j + dm.hopStart];
        }

    }
    
    free(atoms_h);
    free(bonds_h);
    free(angles_h);
    free(dihedrals_h);
    free(hops_h);
    free(dMolec_h);
    
}

#ifdef DEBUG

__global__ void testCalcCharge(Atom *atoms1, Atom *atoms2, double *answers, Environment *enviro){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < enviro->numOfAtoms){
        answers[idx] = calcCharge(atoms1[idx], atoms2[idx], enviro);
    }
}

__global__ void testGetMoleculeFromID(Atom *atoms, DeviceMolecule *molecules,
        Environment enviros, int numberOfTests, int *answers){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < numberOfTests){
        answers[idx] = getMoleculeFromAtomID(atoms[idx], molecules,
                enviros);
    }
    
}

__global__ void testGetFValue(Atom *atom1List, Atom *atom2List, 
        DeviceMolecule *molecules, Environment *enviro, double *fValues, int numberOfTests, Hop *dev_hops){ 
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numberOfTests){
        fValues[idx] = getFValue(atom1List[idx], atom2List[idx], molecules, enviro, dev_hops);
    }
}

__global__ void testCalcBlending(double *d1, double *d2, double *answers, int numberOfTests){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < numberOfTests){
        answers[idx] = calcBlending(d1[idx], d2[idx]);
    }
}

__global__ void testMakePeriodicKernel(double *x, double *box, int n){ 
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n){
        x[idx] = makePeriodic(x[idx], *box);
    }   
}

__global__ void testGetYKernel(int *xValues, int *yValues, int n){ 
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < n){
        yValues[idx] = getYFromIndex(xValues[idx], idx);
    }
}

__global__ void testGetXKernel(int *xValues, int n){
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < n){
        xValues[idx] = getXFromIndex(idx); 
    }
}

__global__ void testCalcLJ(Atom *atoms, Environment *enviro, double *energy){
    Atom atom1 = atoms[0];
    Atom atom2 = atoms[1];

    double testEnergy = calc_lj(atom1, atom2, *enviro);
    
    *energy = testEnergy;
}

#endif //DEBUG
