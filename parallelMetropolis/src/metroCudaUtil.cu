#include "hip/hip_runtime.h"
#include "metroCudaUtil.cuh"


//calculates X (larger indexed atom) for energy calculation based on index in atom array
__device__ int getXFromIndex(int idx){
    int c = -2 * idx;
    int discriminant = 1 - 4 * c;
    int qv = (-1 + sqrtf(discriminant)) / 2;
    return qv + 1;
}

//calculates Y (smaller indexed atom) for energy calculation based on index in atom array
__device__ int getYFromIndex(int x, int idx){
    return idx - (x * x - x) / 2;
}

//apply periodic boundaries
__device__ double makePeriodic(double x, double box){
    
    while(x < -0.5 * box){
        x += box;
    }

    while(x > 0.5 * box){
        x -= box;
    }

    return x;

}

//keep coordinates with box
double wrapBox(double x, double box){

    while(x >  box){
        x -= box;
    }
    while(x < 0){
        x += box;
    }

    return x;
}

void keepMoleculeInBox(Molecule *molecule, Environment *enviro){

    double maxX = DBL_MIN;
    double maxY = DBL_MIN;
    double maxZ = DBL_MIN;

    double minX = DBL_MAX;
    double minY = DBL_MAX;
    double minZ = DBL_MAX;

    double nudge = pow(10.0, -15.0);

    //determine extreme boundaries for molecule
    for (int i = 0; i < molecule->numOfAtoms; i++){
        double currentX = molecule->atoms[i].x;
        double currentY = molecule->atoms[i].y;
        double currentZ = molecule->atoms[i].z;

        if (currentX > maxX)
           maxX = currentX;
        else if (currentX < minX)
           minX = currentX;

        if (currentY > maxY)
            maxY = currentY;
        else if (currentY < minY)
            minY = currentY;

        if (currentZ > maxZ)
            maxZ = currentZ;
        else if (currentZ < minZ)
            minZ = currentZ;
    
    }

    bool isFullyOutX = (minX > enviro->x || maxX < 0) ? true : false;
    bool isFullyOutY = (minY > enviro->y || maxY < 0) ? true : false;
    bool isFullyOutZ = (minZ > enviro->z || maxZ < 0) ? true : false;


    //for each axis, determine if the molecule escapes the environment 
    //and wrap it around into the environment
    for (int i = 0; i < molecule->numOfAtoms; i++){
        double* currentX = &(molecule->atoms[i].x);
        double* currentY = &(molecule->atoms[i].y);
        double* currentZ = &(molecule->atoms[i].z);
        if (maxX > enviro->x){
            if (!isFullyOutX){
                *currentX += (enviro->x - minX);
            }
            *currentX = wrapBox(*currentX + nudge, enviro->x);
        }
        else if (minX < 0){
            if (!isFullyOutX)
                *currentX -= maxX;
            *currentX = wrapBox(*currentX - nudge, enviro->x);
        }

        if (maxY > enviro->y){
            if (!isFullyOutY)
                *currentY += (enviro->y - minY);
            *currentY = wrapBox(*currentY + nudge, enviro->y);
        }
        else if (minY < 0){
            if (!isFullyOutY)
                *currentY -= maxY;
            *currentY = wrapBox(*currentY - nudge, enviro->y);
        }

        if (maxZ > enviro->z){
            if (!isFullyOutZ)
                *currentZ += (enviro->z - minZ);
            *currentZ = wrapBox(*currentZ + nudge, enviro->z);
        }
        else if (minZ < 0){
            if (!isFullyOutZ)
                *currentZ -= maxZ;
            *currentZ = wrapBox(*currentZ - nudge, enviro->z);
        }
        
    }

}
//calculate Lennard-Jones energy between two atoms
__device__ double calc_lj(Atom atom1, Atom atom2, Environment enviro){
    //store LJ constants locally
    double sigma = calcBlending(atom1.sigma, atom2.sigma);
    double epsilon = calcBlending(atom1.epsilon, atom2.epsilon);
    
    //calculate difference in coordinates
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;

    //calculate distance between atoms
    deltaX = makePeriodic(deltaX, enviro.x);
    deltaY = makePeriodic(deltaY, enviro.y);
    deltaZ = makePeriodic(deltaZ, enviro.z);

    const double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);

    //calculate terms
    const double sig2OverR2 = pow(sigma, 2) / r2;
    const double sig6OverR6 = pow(sig2OverR2, 3);
    const double sig12OverR12 = pow(sig6OverR6, 2);
    const double energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);
    
    if (r2 == 0){
        return 0.0;
    }
    else{
        return energy;
    }
}

__global__ void assignAtomPositions(double *dev_doublesX, double *dev_doublesY, double *dev_doublesZ, Atom *atoms, Environment *enviro){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    //for each atom...
    if (idx < enviro->numOfAtoms){
        atoms[idx].x = dev_doublesX[idx] * enviro->x + atoms[idx].x;
        atoms[idx].y = dev_doublesY[idx] * enviro->y + atoms[idx].y;
        atoms[idx].z = dev_doublesZ[idx] * enviro->z + atoms[idx].z;
    }
}

//generate coordinate data for the atoms
void generatePoints(Atom *atoms, Environment *enviro){
    //setup CUDA storage
    hiprandGenerator_t generator;
    double *devXDoubles;
    double *devYDoubles;
    double *devZDoubles;
    //double *hostDoubles;
    Atom *devAtoms;
    Environment *devEnviro;
    
    //hostDoubles = (double *) malloc(sizeof(double) * N);

    //allocate memory on device
    hipMalloc((void**)&devXDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devYDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devZDoubles, enviro->numOfAtoms * sizeof(double));
    hipMalloc((void**)&devAtoms, enviro->numOfAtoms * sizeof(Atom));
    hipMalloc((void**)&devEnviro, sizeof(Environment));

    //copy local data to device
    hipMemcpy(devAtoms, atoms, enviro->numOfAtoms * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(devEnviro, enviro, sizeof(Environment), hipMemcpyHostToDevice);

    //generate doubles for all coordinates
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned int) time(NULL));
    hiprandGenerateUniformDouble(generator, devXDoubles, enviro->numOfAtoms);
    hiprandGenerateUniformDouble(generator, devYDoubles, enviro->numOfAtoms);
    hiprandGenerateUniformDouble(generator, devZDoubles, enviro->numOfAtoms);

    //calculate number of blocks required
    int numOfBlocks = enviro->numOfAtoms / THREADS_PER_BLOCK + (enviro->numOfAtoms % THREADS_PER_BLOCK == 0 ? 0 : 1);

    //assign the doubles to the coordinates
    assignAtomPositions <<< numOfBlocks, THREADS_PER_BLOCK >>> (devXDoubles, devYDoubles, devZDoubles, devAtoms, devEnviro);

    //copy the atoms back to host
    hipMemcpy(atoms, devAtoms, enviro->numOfAtoms * sizeof(Atom), hipMemcpyDeviceToHost);

    //cleanup
    hiprandDestroyGenerator(generator);
    hipFree(devXDoubles);
    hipFree(devYDoubles);
    hipFree(devZDoubles);
    hipFree(devAtoms);
    hipFree(devEnviro);
}

//generate coordinate data for the atoms with all molecules
void generatePoints(Molecule *molecules, Environment *enviro){
    srand(time(NULL));

    for (int i = 0; i < enviro->numOfMolecules; i++){
        double baseX = ( (double) rand() / RAND_MAX) * enviro->x;
        double baseY = ( (double) rand() / RAND_MAX) * enviro->y;
        double baseZ = ( (double) rand() / RAND_MAX) * enviro->z;
        for (int j = 0; j < molecules[i].numOfAtoms; j++){
            molecules[i].atoms[j].x += baseX;
            molecules[i].atoms[j].y += baseY;
            molecules[i].atoms[j].z += baseZ;
        }

        keepMoleculeInBox(&(molecules[i]), enviro);
    }
}

//Calculates the energy of system using molecules
double calcEnergyWrapper(Molecule *molecules, Environment *enviro){
    
    Atom *atoms = (Atom *) malloc(sizeof(Atom) * enviro->numOfAtoms);
    int atomIndex = 0;
    for(int i = 0; i < enviro->numOfMolecules; i++){
        Molecule currentMolecule = molecules[i];
        for(int j = 0; j < currentMolecule.numOfAtoms; j++){
            atoms[atomIndex] = currentMolecule.atoms[j];
            //printf("%d, %f, %f, %f, %f, %f\n", atoms[atomIndex].id, atoms[atomIndex].x,
            //        atoms[atomIndex].y, atoms[atomIndex].z, atoms[atomIndex].sigma,
             //       atoms[atomIndex].epsilon);
            atomIndex++;
        }
    }

    return calcEnergyWrapper(atoms, enviro, molecules);
}

double calcEnergyWrapper(Atom *atoms, Environment *enviro, Molecule *molecules){
    //setup CUDA storage
    double totalEnergy = 0.0;
    Atom *atoms_device;
    double *energySum_device;
    double *energySum_host;
    Environment *enviro_device;

    //calculate CUDA thread mgmt
    int N =(int) ( pow( (float) enviro->numOfAtoms,2)-enviro->numOfAtoms)/2;
    int blocks = N / THREADS_PER_BLOCK + (N % THREADS_PER_BLOCK == 0 ? 0 : 1); 

    //The number of bytes of shared memory per block of
    size_t sharedSize = sizeof(double) * THREADS_PER_BLOCK;
    size_t atomSize = enviro->numOfAtoms * sizeof(Atom);
    size_t energySumSize = N * sizeof(double);
    
    //allocate memory on the device
    energySum_host = (double *) malloc(energySumSize);
    hipMalloc((void **) &atoms_device, atomSize);
    hipMalloc((void **) &energySum_device, energySumSize);
    hipMalloc((void **) &enviro_device, sizeof(Environment));

    //copy data to the device
    hipMemcpy(atoms_device, atoms, atomSize, hipMemcpyHostToDevice);
    hipMemcpy(enviro_device, enviro, sizeof(Environment), hipMemcpyHostToDevice);

    calcEnergy <<<blocks, THREADS_PER_BLOCK>>>(atoms_device, enviro_device, energySum_device);
    
    hipMemcpy(energySum_host, energySum_device, energySumSize, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){

        int c = -2 * i;
        int discriminant = 1 - 4 * c;
        int qv = (-1 + sqrtf(discriminant)) / 2;
        int atomXid = qv + 1;
        
        int atomYid =  i - (atomXid * atomXid - atomXid) / 2;

        if (isnan(energySum_host[i]) != 0 || isinf(energySum_host[i]) != 0){
            energySum_host[i] = calcEnergyOnHost(atoms[atomXid], atoms[atomYid], enviro);
        }
        
        //cout << "EnergySum << " << energySum_host[i] << endl;

        if (molecules != NULL){
            energySum_host[i] = energySum_host[i] * getFValueHost(atoms[atomXid], atoms[atomYid], molecules, enviro); 
        }

        totalEnergy += energySum_host[i];
        //cout << "totalEnergy: " << totalEnergy << endl;

    }

    //cleanup
    hipFree(atoms_device);
    hipFree(energySum_device);
    free(energySum_host);

    return totalEnergy;
}

double calcEnergyOnHost(Atom atom1, Atom atom2, Environment *enviro){
    const double e = 1.602176565 * pow(10.f,-19.f);

    double sigma = sqrt(atom1.sigma * atom2.sigma);
    double epsilon = sqrt(atom1.epsilon * atom2.epsilon);
    
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;
  
    deltaX = make_periodic(deltaX, enviro->x);
    deltaY = make_periodic(deltaY, enviro->y);
    deltaZ = make_periodic(deltaZ, enviro->z);

    double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);

    double r = sqrt(r2);

    double sig2OverR2 = pow(sigma, 2) / r2;
    double sig6OverR6 = pow(sig2OverR2, 3);
    double sig12OverR12 = pow(sig6OverR6, 2);
    double lj_energy = 4.0 * epsilon * (sig12OverR12 - sig6OverR6);

    double charge_energy = (atom2.charge * atom1.charge * pow(e,2) / r);

    double fValue = 1.0; //TODO: make this right
    
    if (r2 == 0.0){
        lj_energy = 0.0;
        charge_energy = 0.0;
    }

    return fValue * (lj_energy + charge_energy);

}

__global__ void calcEnergy(Atom *atoms, Environment *enviro, double *energySum){

//need to figure out how many threads per block will be executed
// must be a power of 2
    __shared__ double cache[THREADS_PER_BLOCK];

    int cacheIndex = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double lj_energy,charge_energy, fValue;

    int N =(int) ( pow( (float) enviro->numOfAtoms,2)-enviro->numOfAtoms)/2;

    if(idx < N ){
    //calculate the x and y positions in the Atom array
        int xAtom_pos, yAtom_pos;
        xAtom_pos = getXFromIndex(idx);
        yAtom_pos = getYFromIndex(xAtom_pos, idx);

        Atom xAtom, yAtom;
        xAtom = atoms[xAtom_pos];
        yAtom = atoms[yAtom_pos];

        if(xAtom.sigma < 0 || xAtom.epsilon < 0 || yAtom.sigma < 0 || yAtom.epsilon < 0){
            energySum[idx] = 0.0;
        }
        else{
            lj_energy = calc_lj(xAtom,yAtom,*enviro);
            charge_energy = calcCharge(xAtom, yAtom, enviro);
            fValue = 1.0; //TODO: Fix after fValue calculation is moved to device
            
            energySum[idx] = fValue * (lj_energy + charge_energy);
        }
    }
    else {
        energySum[idx] = 0.0;
    }


    /**
// set the cache values
cache[cacheIndex] = lj_energy;
// synchronize threads in this block
__syncthreads();
// adds 2 positions together
int i = blockDim.x/2;
while (i != 0) {
if (cacheIndex < i)
cache[cacheIndex] += cache[cacheIndex + i];
__syncthreads();
i /= 2;
}
// copy this block's sum to the enrgySums array
// at its block index postition
if (cacheIndex == 0)
energySum[blockIdx.x] = cache[0];
*/

}

__device__ double calcCharge(Atom atom1, Atom atom2, Environment *enviro){
    const double e = 1.602176565 * pow(10.f,-19.f);
 
    //calculate difference in coordinates
    double deltaX = atom1.x - atom2.x;
    double deltaY = atom1.y - atom2.y;
    double deltaZ = atom1.z - atom2.z;

    //calculate distance between atoms
    deltaX = makePeriodic(deltaX, enviro->x);
    deltaY = makePeriodic(deltaY, enviro->y);
    deltaZ = makePeriodic(deltaZ, enviro->z);

    double r2 = (deltaX * deltaX) +
                      (deltaY * deltaY) + 
                      (deltaZ * deltaZ);
    
    double r = sqrt(r2);


    if (r == 0.0){
        return 0.0;
    }
    else{
        return (atom1.charge * atom2.charge * pow(e,2) / r);
    }
}

__device__ double calcBlending(double d1, double d2){
    return sqrt(d1 * d2);
}

//returns the molecule that contains a given atom
__device__ int getMoleculeFromAtomID(Atom a1, Molecule *molecules, Environment enviro){
    int atomId = a1.id;
    int currentIndex = enviro.numOfMolecules - 1;
    int molecId = molecules[currentIndex].id;
    while(atomId < molecId && currentIndex > 0){
        currentIndex -= 1;
        molecId = molecules[currentIndex].id;
    }
    return molecId;

}

__device__ double getFValue(Atom atom1, Atom atom2, Molecule *molecules, Environment *enviro){
    int m1 = getMoleculeFromAtomID(atom1, molecules, *enviro);
    int m2 = getMoleculeFromAtomID(atom2, molecules, *enviro);
    Molecule molec = molecules[0];
    for(int i = 0; i < enviro->numOfMolecules; i++){
        if(molecules[i].id == m1){
            molec = molecules[i];
            break;
        }
    }

    if(m1 != m2)
        return 1.0;
	 else if( hopGE3(atom1.id, atom2.id, molecules[m1]) )     
		  return 0.5;
	 else
		  return 0.0;
}

__device__ int hopGE3(int atom1, int atom2, Molecule molecule){
    for(int x=0; x< molecule.numOfHops; x++){
		      Hop myHop = molecule.hops[x];
				if(myHop.atom1==atom1 && myHop.atom2==atom2)
				    return 1;
	 }
	 return 0;
}

//returns the molecule that contains a given atom
Molecule* getMoleculeFromAtomIDHost(Atom a1, Molecule *molecules, Environment enviro){
    int atomId = a1.id;
    int currentIndex = enviro.numOfMolecules - 1;
    int molecId = molecules[currentIndex].id;
    while(atomId < molecId && currentIndex > 0){
        currentIndex -= 1;
        molecId = molecules[currentIndex].id;
    }
    return &molecules[currentIndex];

}

double getFValueHost(Atom atom1, Atom atom2, Molecule *molecules, Environment *enviro){
    Molecule *m1 = getMoleculeFromAtomIDHost(atom1, molecules, *enviro);
    Molecule *m2 = getMoleculeFromAtomIDHost(atom2, molecules, *enviro);
    Molecule molec = molecules[0];
    for(int i = 0; i < enviro->numOfMolecules; i++){
        if(molecules[i].id == m1->id){
            molec = molecules[i];
            break;
        }
    }

    if(m1->id != m2->id)
        return 1.0;
	 else if(hopGE3Host(atom1.id, atom2.id, *m1) == 1)     
		  return 0.5;
	 else
		  return 0.0;
}

int hopGE3Host(int atom1, int atom2, Molecule molecule){
    for(int x=0; x< molecule.numOfHops; x++){
		      Hop myHop = molecule.hops[x];
				if((myHop.atom1==atom1 && myHop.atom2==atom2) ||
                        (myHop.atom1 == atom2 && myHop.atom2 == atom1) )
				    return 1;
	 }
	 return 0;
}

void rotateMolecule(Molecule molecule, Atom pivotAtom, double maxRotation){
    //save pivot atom coordinates because they will change
    double pivotAtomX = pivotAtom.x;
    double pivotAtomY = pivotAtom.y;
    double pivotAtomZ = pivotAtom.z;

    //translate entire molecule to place pivotAtom at origin
    for (int i = 0; i < molecule.numOfAtoms; i++){
        molecule.atoms[i].x -= pivotAtomX;
        molecule.atoms[i].y -= pivotAtomY;
        molecule.atoms[i].z -= pivotAtomZ;
    }

    srand(time(NULL));
    double dtr = PI / 180.0;

    //rotate molecule about origin
    for (int axis = 0; axis < 3; axis++){
        double rotation = ((double) rand() / (double) RAND_MAX) * maxRotation * dtr;
        double sinrot = sin(rotation);
        double cosrot = cos(rotation);
        if (axis == 0){ //rotate about x-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldY = thisAtom->y;
                double oldZ = thisAtom->z;
                thisAtom->y = cosrot * oldY + sinrot * oldZ;
                thisAtom->z = cosrot * oldZ - sinrot * oldY;
            }
        }
        else if (axis == 1){ //rotate about y-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldX = thisAtom->x;
                double oldZ = thisAtom->z;
                thisAtom->x = cosrot * oldX - sinrot * oldZ;
                thisAtom->z = cosrot * oldZ + sinrot * oldX;
            }
        }
        if (axis == 2){ //rotate about z-axis
            for (int i = 0; i < molecule.numOfAtoms; i++){
                Atom *thisAtom = &(molecule.atoms[i]);
                double oldX = thisAtom->x;
                double oldY = thisAtom->y;
                thisAtom->x = cosrot * oldX + sinrot * oldY;
                thisAtom->y = cosrot * oldY - sinrot * oldX;
            }
        }
    }

    //translate entire molecule back based on original pivot point
    for (int i = 0; i < molecule.numOfAtoms; i++){
        molecule.atoms[i].x += pivotAtomX;
        molecule.atoms[i].y += pivotAtomY;
        molecule.atoms[i].z += pivotAtomZ;
    }
}

/**
  This  is currently a stub pending information from Dr. Acevedo
*/
double solventAccessibleSurfaceArea(){
    return -1.f;
}

/**
  This is currently a stub pending information from Dr. Acevedo
*/
double soluteSolventDistributionFunction(){
    return -1.f;
}

/**
  This is currently a stub pending information from Dr. Acevedo
*/
double atomAtomDistributionFunction(){
    return -1.f;
}

/**
  This is currently a stub pending information from Dr. Acevedo
*/
double solventSolventTotalEnergy(){
    return -1.f;
}

/**
  This is currently a stub pending information from Dr. Acevedo
*/
double soluteSolventTotalEnergy(){
    return -1.f;
}


#ifdef DEBUG

//these are all test wrappers for __device__ functions because they cannot be called from an outside source file.

__global__ void testCalcCharge(Atom *atoms1, Atom *atoms2, double *answers, Environment *enviro){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < enviro->numOfAtoms){
        answers[idx] = calcCharge(atoms1[idx], atoms2[idx], enviro);
    }
}

__global__ void testGetMoleculeFromID(Atom *atoms, Molecule *molecules,
        Environment enviros, int numberOfTests, int *answers){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < numberOfTests){
        answers[idx] = getMoleculeFromAtomID(atoms[idx], molecules,
                enviros);
    }
    
}

__global__ void testGetFValue(Atom *atom1List, Atom *atom2List, 
        Molecule *molecules, Environment *enviro, double *fValues, int numberOfTests){ 
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numberOfTests){
        fValues[idx] = getFValue(atom1List[idx], atom2List[idx], molecules, enviro);
    }
}

__global__ void testCalcBlending(double *d1, double *d2, double *answers, int numberOfTests){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < numberOfTests){
        answers[idx] = calcBlending(d1[idx], d2[idx]);
    }
}

__global__ void testMakePeriodicKernel(double *x, double *box, int n){ 
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n){
        x[idx] = makePeriodic(x[idx], *box);
    }   
}

__global__ void testGetYKernel(int *xValues, int *yValues, int n){ 
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < n){
        yValues[idx] = getYFromIndex(xValues[idx], idx);
    }
}

__global__ void testGetXKernel(int *xValues, int n){
    int idx =  threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < n){
        xValues[idx] = getXFromIndex(idx); 
    }
}

__global__ void testCalcLJ(Atom *atoms, Environment *enviro, double *energy){
    Atom atom1 = atoms[0];
    Atom atom2 = atoms[1];

    double testEnergy = calc_lj(atom1, atom2, *enviro);
    
    *energy = testEnergy;
}

#endif //DEBUG
