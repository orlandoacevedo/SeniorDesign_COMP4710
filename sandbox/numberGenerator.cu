#include "hip/hip_runtime.h"
#includes <hip/hip_runtime.h>
#includes <hiprand/hiprand_kernel.h>


__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
        int id = threadIdx.x;
        hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState ) 
{
        int ind = threadIdx.x;
        hiprandState localState = globalState[ind];
        float RANDOM = hiprand_uniform( &localState );
        globalState[ind] = localState; 
}

int main( int argc, char** argv) 
{
        dim3 tpb(N,1,1); 
        hiprandState* devStates;
        hipMalloc ( &devStates, N*sizeof( hiprandState ) );
                    
        // setup seeds
        setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );

        // generate random numbers
        generate <<< 1, tpb >>> ( devStates );

        return 0;
}
