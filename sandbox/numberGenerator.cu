#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
        int id = threadIdx.x;
        hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState, double* dev_storage ) 
{
        int ind = threadIdx.x;
        hiprandState localState = globalState[ind];
        double stable_storage = hiprand_uniform_double( &localState );
        dev_storage = &stable_storage;
        globalState[ind] = localState; 
}

int main( int argc, char** argv) 
{ 
        int N = 1;
        hiprandState* devStates;
        hipMalloc ( &devStates, N*sizeof( hiprandState ) );
        
        double *host_storage; 
        double *dev_storage;

        host_storage = (double *) malloc(sizeof(double));

        *host_storage = -1.0;
        hipMalloc(&dev_storage, sizeof(*dev_storage));

        // setup seeds
        setup_kernel <<< 1, 1 >>> ( devStates, time(NULL) );

        // generate random numbers
        generate <<< 1, 1 >>> ( devStates, dev_storage );

        hipMemcpy(host_storage, dev_storage, sizeof(*host_storage), hipMemcpyDeviceToHost);

        printf("host_storage = %f", *host_storage);

        return 0;
}
