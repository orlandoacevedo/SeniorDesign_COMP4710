#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
        int id = threadIdx.x;
        hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState, float* dev_storage ) 
{
        int ind = threadIdx.x;
        hiprandState localState = globalState[ind];
        float stable_storage = hiprand_uniform( &globalState )[ind];
        dev_storage = &stable_storage;
        globalState[ind] = localState; 
}

int main( int argc, char** argv) 
{ 
        int N = 1;
        hiprandState* devStates;
        hipMalloc ( &devStates, N*sizeof( hiprandState ) );
        
        float *host_storage; 
        float *dev_storage;

        host_storage = (float *) malloc(sizeof(float));

        *host_storage = -1.0;
        printf("host_storage before = %f\n", *host_storage);
        hipMalloc(&dev_storage, sizeof(*dev_storage));

        // setup seeds
        setup_kernel <<< 1, 1 >>> ( devStates, time(NULL) );

        // generate random numbers
        generate <<< 1, 1 >>> ( devStates, dev_storage );

        hipMemcpy(host_storage, dev_storage, sizeof(*host_storage), hipMemcpyDeviceToHost);

        printf("host_storage = %f\n", *host_storage);

        return 0;
}
