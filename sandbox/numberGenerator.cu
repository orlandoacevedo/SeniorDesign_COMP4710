#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__ void generate( hiprandState* globalState, double* dev_storage ) 
{
        int idx = threadIdx.x + blockDim.x * blockIdx.x;

        hiprand_init ( 1234, idx, 0, &globalState[idx] );

        hiprandState localState = globalState[idx];
        double stable_storage = hiprand_uniform_double( &globalState[idx] );
        dev_storage = &stable_storage;
        globalState[idx] = localState; 
}

int main( int argc, char** argv) 
{ 
        int N = 1;
        hiprandState* devStates;
        hipMalloc ((void**) &devStates, N*sizeof( hiprandState ) );
        
        double *host_storage; 
        double *dev_storage;

        host_storage = (double *) malloc(sizeof(double));

        *host_storage = -1.0;
        printf("host_storage before = %f\n", *host_storage);
        
        hipMalloc((void**) &dev_storage, sizeof(double));

        // generate random numbers
        generate <<< 1, 1 >>> ( devStates, dev_storage );

        hipMemcpy(host_storage, dev_storage, sizeof(double), hipMemcpyDeviceToHost);

        printf("host_storage = %f\n", *host_storage);

        return 0;
}
