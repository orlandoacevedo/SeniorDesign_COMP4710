#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

main(){
    int i, n = 100;
    hiprandGenerator_t gen;
    double *devData, *hostData;

    hostData = (double *)calloc(n, sizeof(double));

    hipMalloc((void **)&devData, n * sizeof(double));

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    hiprandGenerateUniformDouble(gen, devData, n);

    hipMemcpy(hostData, devData, n * sizeof(double), hipMemcpyDeviceToHost);

    for (i = 0; i < n; i++) {
        printf("%1.4f\n", hostData[i]);
    }

    hiprandDestroyGenerator(gen);
    hipFree(devData);
    free(hostData);

    return 0;
}
